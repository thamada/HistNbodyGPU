#include "hip/hip_runtime.h"
#include "cutil.h"

__device__ float4 
inter(float4 xj, float4 xi, float4 apot)
{
  float mj    = xj.w;        // Mass Mj
  float ieps2 = xi.w;        // epsilon^2
  float dx = xj.x - xi.x;    // Coordinates Xj - Xi
  float dy = xj.y - xi.y;    // Coordinates Yj - Yi
  float dz = xj.z - xi.z;    // Coordinates Zj - Zi
  float r2 = (dx*dx+ieps2)+dy*dy+dz*dz;
  float r1i = 1/sqrt(r2);
  float r2i = r1i * r1i;
  float mr3i = mj * r2i * r1i;
  apot.x += dx * mr3i;       // Accel AXi
  apot.y += dy * mr3i;       // Accel AYi
  apot.z += dz * mr3i;       // Accel AZi
  return (apot);
}
#define NTHRE (128) // blockDim.x��Ʊ��
__global__ void
kernel(float4* g_xj,
       float* g_xi,
       float* g_fi,
       int ni,
       int nj)
{
  int tid      = threadIdx.x;
  int i = blockIdx.x*NTHRE+tid;
  float4 ai = make_float4(0.0, 0.0, 0.0, 0.0);
  float4 xi;
  xi.x = g_xi[i];
  xi.y = g_xi[i+ni];
  xi.z = g_xi[i+ni*2];
  xi.w = g_xi[i+ni*3];
  __shared__ float4 s_xj[NTHRE];
  for(int j = 0; j<nj; j+=NTHRE){
    __syncthreads();
    s_xj[tid] = g_xj[j+tid];
    __syncthreads();
    ai = inter(s_xj[0], xi, ai); ai = inter(s_xj[1], xi, ai); ai = inter(s_xj[2], xi, ai); ai = inter(s_xj[3], xi, ai);
    ai = inter(s_xj[4], xi, ai); ai = inter(s_xj[5], xi, ai); ai = inter(s_xj[6], xi, ai); ai = inter(s_xj[7], xi, ai);
    ai = inter(s_xj[8], xi, ai); ai = inter(s_xj[9], xi, ai);

    ai = inter(s_xj[10], xi, ai); ai = inter(s_xj[11], xi, ai); ai = inter(s_xj[12], xi, ai); ai = inter(s_xj[13], xi, ai);
    ai = inter(s_xj[14], xi, ai); ai = inter(s_xj[15], xi, ai); ai = inter(s_xj[16], xi, ai); ai = inter(s_xj[17], xi, ai);
    ai = inter(s_xj[18], xi, ai); ai = inter(s_xj[19], xi, ai);

    ai = inter(s_xj[20], xi, ai); ai = inter(s_xj[21], xi, ai); ai = inter(s_xj[22], xi, ai); ai = inter(s_xj[23], xi, ai);
    ai = inter(s_xj[24], xi, ai); ai = inter(s_xj[25], xi, ai); ai = inter(s_xj[26], xi, ai); ai = inter(s_xj[27], xi, ai);
    ai = inter(s_xj[28], xi, ai); ai = inter(s_xj[29], xi, ai);

    ai = inter(s_xj[30], xi, ai); ai = inter(s_xj[31], xi, ai); ai = inter(s_xj[32], xi, ai); ai = inter(s_xj[33], xi, ai);
    ai = inter(s_xj[34], xi, ai); ai = inter(s_xj[35], xi, ai); ai = inter(s_xj[36], xi, ai); ai = inter(s_xj[37], xi, ai);
    ai = inter(s_xj[38], xi, ai); ai = inter(s_xj[39], xi, ai);

    ai = inter(s_xj[40], xi, ai); ai = inter(s_xj[41], xi, ai); ai = inter(s_xj[42], xi, ai); ai = inter(s_xj[43], xi, ai);
    ai = inter(s_xj[44], xi, ai); ai = inter(s_xj[45], xi, ai); ai = inter(s_xj[46], xi, ai); ai = inter(s_xj[47], xi, ai);
    ai = inter(s_xj[48], xi, ai); ai = inter(s_xj[49], xi, ai);

    ai = inter(s_xj[50], xi, ai); ai = inter(s_xj[51], xi, ai); ai = inter(s_xj[52], xi, ai); ai = inter(s_xj[53], xi, ai);
    ai = inter(s_xj[54], xi, ai); ai = inter(s_xj[55], xi, ai); ai = inter(s_xj[56], xi, ai); ai = inter(s_xj[57], xi, ai);
    ai = inter(s_xj[58], xi, ai); ai = inter(s_xj[59], xi, ai);

    ai = inter(s_xj[60], xi, ai); ai = inter(s_xj[61], xi, ai); ai = inter(s_xj[62], xi, ai); ai = inter(s_xj[63], xi, ai);
    ai = inter(s_xj[64], xi, ai); ai = inter(s_xj[65], xi, ai); ai = inter(s_xj[66], xi, ai); ai = inter(s_xj[67], xi, ai);
    ai = inter(s_xj[68], xi, ai); ai = inter(s_xj[69], xi, ai);

    ai = inter(s_xj[70], xi, ai); ai = inter(s_xj[71], xi, ai); ai = inter(s_xj[72], xi, ai); ai = inter(s_xj[73], xi, ai);
    ai = inter(s_xj[74], xi, ai); ai = inter(s_xj[75], xi, ai); ai = inter(s_xj[76], xi, ai); ai = inter(s_xj[77], xi, ai);
    ai = inter(s_xj[78], xi, ai); ai = inter(s_xj[79], xi, ai);

    ai = inter(s_xj[80], xi, ai); ai = inter(s_xj[81], xi, ai); ai = inter(s_xj[82], xi, ai); ai = inter(s_xj[83], xi, ai);
    ai = inter(s_xj[84], xi, ai); ai = inter(s_xj[85], xi, ai); ai = inter(s_xj[86], xi, ai); ai = inter(s_xj[87], xi, ai);
    ai = inter(s_xj[88], xi, ai); ai = inter(s_xj[89], xi, ai);

    ai = inter(s_xj[90], xi, ai); ai = inter(s_xj[91], xi, ai); ai = inter(s_xj[92], xi, ai); ai = inter(s_xj[93], xi, ai);
    ai = inter(s_xj[94], xi, ai); ai = inter(s_xj[95], xi, ai); ai = inter(s_xj[96], xi, ai); ai = inter(s_xj[97], xi, ai);
    ai = inter(s_xj[98], xi, ai); ai = inter(s_xj[99], xi, ai);

    ai = inter(s_xj[100], xi, ai); ai = inter(s_xj[101], xi, ai); ai = inter(s_xj[102], xi, ai); ai = inter(s_xj[103], xi, ai);
    ai = inter(s_xj[104], xi, ai); ai = inter(s_xj[105], xi, ai); ai = inter(s_xj[106], xi, ai); ai = inter(s_xj[107], xi, ai);
    ai = inter(s_xj[108], xi, ai); ai = inter(s_xj[109], xi, ai);

    ai = inter(s_xj[110], xi, ai); ai = inter(s_xj[111], xi, ai); ai = inter(s_xj[112], xi, ai); ai = inter(s_xj[113], xi, ai);
    ai = inter(s_xj[114], xi, ai); ai = inter(s_xj[115], xi, ai); ai = inter(s_xj[116], xi, ai); ai = inter(s_xj[117], xi, ai);
    ai = inter(s_xj[118], xi, ai); ai = inter(s_xj[119], xi, ai);

    ai = inter(s_xj[120], xi, ai); ai = inter(s_xj[121], xi, ai); ai = inter(s_xj[122], xi, ai); ai = inter(s_xj[123], xi, ai);
    ai = inter(s_xj[124], xi, ai); ai = inter(s_xj[125], xi, ai); ai = inter(s_xj[126], xi, ai); ai = inter(s_xj[127], xi, ai);
  }
  if(i<ni){
    g_fi[i]      = ai.x;
    g_fi[i+ni]   = ai.y;
    g_fi[i+ni*2] = ai.z;
  }
}

#include "api.h"
