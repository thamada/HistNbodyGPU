#include "hip/hip_runtime.h"
#include "cutil.h"

__device__ float4 
inter(float4 xj, float4 xi)
{
  float mj    = xj.w;        // Mass Mj
  float ieps2 = xi.w;        // epsilon^2
  float dx = xj.x - xi.x;    // Coordinates Xj - Xi
  float dy = xj.y - xi.y;    // Coordinates Yj - Yi
  float dz = xj.z - xi.z;    // Coordinates Zj - Zi
  float r2 = (dx*dx+ieps2)+dy*dy+dz*dz;
  float r1i = 1/sqrt(r2);
  float r2i = r1i * r1i;
  float mr3i = mj * r2i * r1i;

  float4 output;
  output.x = dx;
  output.y = dy;
  output.z = dz;
  output.w = mr3i;
  return (output);
  /*
  apot.x += dx * mr3i;       // Accel AXi
  apot.y += dy * mr3i;       // Accel AYi
  apot.z += dz * mr3i;       // Accel AZi
  return (apot);
  */
}
#define NTHRE (128) // blockDim.x��Ʊ��
__global__ void
kernel(float4* g_xj,
       float* g_xi,
       float* g_fi,
       int ni,
       int nj)
{
  int tid = threadIdx.x;
  int i = blockIdx.x*NTHRE+tid;
  float4 ai = make_float4(0.0, 0.0, 0.0, 0.0);
  float4 xi;
  xi.x = g_xi[i];
  xi.y = g_xi[i+ni];
  xi.z = g_xi[i+ni*2];
  xi.w = g_xi[i+ni*3];
  __shared__ float4 s_xj[NTHRE];

  float4 mf000;float4 mf001;float4 mf002;float4 mf003;float4 mf004;float4 mf005;float4 mf006;float4 mf007;float4 mf008;float4 mf009;
  float4 mf010;float4 mf011;float4 mf012;float4 mf013;float4 mf014;float4 mf015;float4 mf016;float4 mf017;float4 mf018;float4 mf019;
  float4 mf020;float4 mf021;float4 mf022;float4 mf023;float4 mf024;float4 mf025;float4 mf026;float4 mf027;float4 mf028;float4 mf029;
  float4 mf030;float4 mf031;float4 mf032;float4 mf033;float4 mf034;float4 mf035;float4 mf036;float4 mf037;float4 mf038;float4 mf039;
  float4 mf040;float4 mf041;float4 mf042;float4 mf043;float4 mf044;float4 mf045;float4 mf046;float4 mf047;float4 mf048;float4 mf049;
  float4 mf050;float4 mf051;float4 mf052;float4 mf053;float4 mf054;float4 mf055;float4 mf056;float4 mf057;float4 mf058;float4 mf059;
  float4 mf060;float4 mf061;float4 mf062;float4 mf063;float4 mf064;float4 mf065;float4 mf066;float4 mf067;float4 mf068;float4 mf069;
  float4 mf070;float4 mf071;float4 mf072;float4 mf073;float4 mf074;float4 mf075;float4 mf076;float4 mf077;float4 mf078;float4 mf079;
  float4 mf080;float4 mf081;float4 mf082;float4 mf083;float4 mf084;float4 mf085;float4 mf086;float4 mf087;float4 mf088;float4 mf089;
  float4 mf090;float4 mf091;float4 mf092;float4 mf093;float4 mf094;float4 mf095;float4 mf096;float4 mf097;float4 mf098;float4 mf099;
  float4 mf100;float4 mf101;float4 mf102;float4 mf103;float4 mf104;float4 mf105;float4 mf106;float4 mf107;float4 mf108;float4 mf109;
  float4 mf110;float4 mf111;float4 mf112;float4 mf113;float4 mf114;float4 mf115;float4 mf116;float4 mf117;float4 mf118;float4 mf119;
  float4 mf120;float4 mf121;float4 mf122;float4 mf123;float4 mf124;float4 mf125;float4 mf126;float4 mf127;

  for(int j = 0; j<nj; j+=NTHRE){
    __syncthreads();
    s_xj[tid] = g_xj[j+tid];
    __syncthreads();
    mf000=inter(s_xj[0], xi);
    mf001=inter(s_xj[1], xi);mf002=inter(s_xj[2], xi);mf003=inter(s_xj[3], xi);mf004=inter(s_xj[4], xi);
    mf005=inter(s_xj[5], xi);mf006=inter(s_xj[6], xi);mf007=inter(s_xj[7], xi);mf008=inter(s_xj[8], xi);
    mf009=inter(s_xj[9], xi);mf010=inter(s_xj[10], xi);mf011=inter(s_xj[11], xi);mf012=inter(s_xj[12], xi);
    mf013=inter(s_xj[13], xi);mf014=inter(s_xj[14], xi);mf015=inter(s_xj[15], xi);mf016=inter(s_xj[16], xi);
    mf017=inter(s_xj[17], xi);mf018=inter(s_xj[18], xi);mf019=inter(s_xj[19], xi);mf020=inter(s_xj[20], xi);
    mf021=inter(s_xj[21], xi);mf022=inter(s_xj[22], xi);mf023=inter(s_xj[23], xi);mf024=inter(s_xj[24], xi);
    mf025=inter(s_xj[25], xi);mf026=inter(s_xj[26], xi);mf027=inter(s_xj[27], xi);mf028=inter(s_xj[28], xi);
    mf029=inter(s_xj[29], xi);mf030=inter(s_xj[30], xi);mf031=inter(s_xj[31], xi);mf032=inter(s_xj[32], xi);
    mf033=inter(s_xj[33], xi);mf034=inter(s_xj[34], xi);mf035=inter(s_xj[35], xi);mf036=inter(s_xj[36], xi);
    mf037=inter(s_xj[37], xi);mf038=inter(s_xj[38], xi);mf039=inter(s_xj[39], xi);mf040=inter(s_xj[40], xi);
    mf041=inter(s_xj[41], xi);mf042=inter(s_xj[42], xi);mf043=inter(s_xj[43], xi);mf044=inter(s_xj[44], xi);
    mf045=inter(s_xj[45], xi);mf046=inter(s_xj[46], xi);mf047=inter(s_xj[47], xi);mf048=inter(s_xj[48], xi);
    mf049=inter(s_xj[49], xi);mf050=inter(s_xj[50], xi);mf051=inter(s_xj[51], xi);mf052=inter(s_xj[52], xi);
    mf053=inter(s_xj[53], xi);mf054=inter(s_xj[54], xi);mf055=inter(s_xj[55], xi);mf056=inter(s_xj[56], xi);
    mf057=inter(s_xj[57], xi);mf058=inter(s_xj[58], xi);mf059=inter(s_xj[59], xi);mf060=inter(s_xj[60], xi);
    mf061=inter(s_xj[61], xi);mf062=inter(s_xj[62], xi);mf063=inter(s_xj[63], xi);mf064=inter(s_xj[64], xi);
    mf065=inter(s_xj[65], xi);mf066=inter(s_xj[66], xi);mf067=inter(s_xj[67], xi);mf068=inter(s_xj[68], xi);
    mf069=inter(s_xj[69], xi);mf070=inter(s_xj[70], xi);mf071=inter(s_xj[71], xi);mf072=inter(s_xj[72], xi);
    mf073=inter(s_xj[73], xi);mf074=inter(s_xj[74], xi);mf075=inter(s_xj[75], xi);mf076=inter(s_xj[76], xi);
    mf077=inter(s_xj[77], xi);mf078=inter(s_xj[78], xi);mf079=inter(s_xj[79], xi);mf080=inter(s_xj[80], xi);
    mf081=inter(s_xj[81], xi);mf082=inter(s_xj[82], xi);mf083=inter(s_xj[83], xi);mf084=inter(s_xj[84], xi);
    mf085=inter(s_xj[85], xi);mf086=inter(s_xj[86], xi);mf087=inter(s_xj[87], xi);mf088=inter(s_xj[88], xi);
    mf089=inter(s_xj[89], xi);mf090=inter(s_xj[90], xi);mf091=inter(s_xj[91], xi);mf092=inter(s_xj[92], xi);
    mf093=inter(s_xj[93], xi);mf094=inter(s_xj[94], xi);mf095=inter(s_xj[95], xi);mf096=inter(s_xj[96], xi);
    mf097=inter(s_xj[97], xi);mf098=inter(s_xj[98], xi);mf099=inter(s_xj[99], xi);mf100=inter(s_xj[100], xi);
    mf101=inter(s_xj[101], xi);mf102=inter(s_xj[102], xi);mf103=inter(s_xj[103], xi);mf104=inter(s_xj[104], xi);
    mf105=inter(s_xj[105], xi);mf106=inter(s_xj[106], xi);mf107=inter(s_xj[107], xi);mf108=inter(s_xj[108], xi);
    mf109=inter(s_xj[109], xi);mf110=inter(s_xj[110], xi);mf111=inter(s_xj[111], xi);mf112=inter(s_xj[112], xi);
    mf113=inter(s_xj[113], xi);mf114=inter(s_xj[114], xi);mf115=inter(s_xj[115], xi);mf116=inter(s_xj[116], xi);
    mf117=inter(s_xj[117], xi);mf118=inter(s_xj[118], xi);mf119=inter(s_xj[119], xi);mf120=inter(s_xj[120], xi);
    mf121=inter(s_xj[121], xi);mf122=inter(s_xj[122], xi);mf123=inter(s_xj[123], xi);mf124=inter(s_xj[124], xi);
    mf125=inter(s_xj[125], xi);mf126=inter(s_xj[126], xi);mf127=inter(s_xj[127], xi);

    ai.x += (mf000.w * mf000.x)+
      (mf001.w * mf001.x)+(mf002.w * mf002.x)+(mf003.w * mf003.x)+(mf004.w * mf004.x)+(mf005.w * mf005.x)+(mf006.w * mf006.x)+(mf007.w * mf007.x)+(mf008.w * mf008.x)+(mf009.w * mf009.x)+
      (mf010.w * mf010.x)+(mf011.w * mf011.x)+(mf012.w * mf012.x)+(mf013.w * mf013.x)+(mf014.w * mf014.x)+(mf015.w * mf015.x)+(mf016.w * mf016.x)+(mf017.w * mf017.x)+(mf018.w * mf018.x)+
      (mf019.w * mf019.x)+(mf020.w * mf020.x)+(mf021.w * mf021.x)+(mf022.w * mf022.x)+(mf023.w * mf023.x)+(mf024.w * mf024.x)+(mf025.w * mf025.x)+(mf026.w * mf026.x)+(mf027.w * mf027.x)+
      (mf028.w * mf028.x)+(mf029.w * mf029.x)+(mf030.w * mf030.x)+(mf031.w * mf031.x)+(mf032.w * mf032.x)+(mf033.w * mf033.x)+(mf034.w * mf034.x)+(mf035.w * mf035.x)+(mf036.w * mf036.x)+
      (mf037.w * mf037.x)+(mf038.w * mf038.x)+(mf039.w * mf039.x)+(mf040.w * mf040.x)+(mf041.w * mf041.x)+(mf042.w * mf042.x)+(mf043.w * mf043.x)+(mf044.w * mf044.x)+(mf045.w * mf045.x)+
      (mf046.w * mf046.x)+(mf047.w * mf047.x)+(mf048.w * mf048.x)+(mf049.w * mf049.x)+(mf050.w * mf050.x)+(mf051.w * mf051.x)+(mf052.w * mf052.x)+(mf053.w * mf053.x)+(mf054.w * mf054.x)+
      (mf055.w * mf055.x)+(mf056.w * mf056.x)+(mf057.w * mf057.x)+(mf058.w * mf058.x)+(mf059.w * mf059.x)+(mf060.w * mf060.x)+(mf061.w * mf061.x)+(mf062.w * mf062.x)+(mf063.w * mf063.x)+
      (mf064.w * mf064.x)+(mf065.w * mf065.x)+(mf066.w * mf066.x)+(mf067.w * mf067.x)+(mf068.w * mf068.x)+(mf069.w * mf069.x)+(mf070.w * mf070.x)+(mf071.w * mf071.x)+(mf072.w * mf072.x)+
      (mf073.w * mf073.x)+(mf074.w * mf074.x)+(mf075.w * mf075.x)+(mf076.w * mf076.x)+(mf077.w * mf077.x)+(mf078.w * mf078.x)+(mf079.w * mf079.x)+(mf080.w * mf080.x)+(mf081.w * mf081.x)+
      (mf082.w * mf082.x)+(mf083.w * mf083.x)+(mf084.w * mf084.x)+(mf085.w * mf085.x)+(mf086.w * mf086.x)+(mf087.w * mf087.x)+(mf088.w * mf088.x)+(mf089.w * mf089.x)+(mf090.w * mf090.x)+
      (mf091.w * mf091.x)+(mf092.w * mf092.x)+(mf093.w * mf093.x)+(mf094.w * mf094.x)+(mf095.w * mf095.x)+(mf096.w * mf096.x)+(mf097.w * mf097.x)+(mf098.w * mf098.x)+(mf099.w * mf099.x)+
      (mf100.w * mf100.x)+(mf101.w * mf101.x)+(mf102.w * mf102.x)+(mf103.w * mf103.x)+(mf104.w * mf104.x)+(mf105.w * mf105.x)+(mf106.w * mf106.x)+(mf107.w * mf107.x)+(mf108.w * mf108.x)+
      (mf109.w * mf109.x)+(mf110.w * mf110.x)+(mf111.w * mf111.x)+(mf112.w * mf112.x)+(mf113.w * mf113.x)+(mf114.w * mf114.x)+(mf115.w * mf115.x)+(mf116.w * mf116.x)+(mf117.w * mf117.x)+
      (mf118.w * mf118.x)+(mf119.w * mf119.x)+(mf120.w * mf120.x)+(mf121.w * mf121.x)+(mf122.w * mf122.x)+(mf123.w * mf123.x)+(mf124.w * mf124.x)+(mf125.w * mf125.x)+(mf126.w * mf126.x)+
      (mf127.w * mf127.x);

    ai.y += (mf000.w * mf000.y)+
      (mf001.w * mf001.y)+(mf002.w * mf002.y)+(mf003.w * mf003.y)+(mf004.w * mf004.y)+(mf005.w * mf005.y)+(mf006.w * mf006.y)+(mf007.w * mf007.y)+(mf008.w * mf008.y)+(mf009.w * mf009.y)+
      (mf010.w * mf010.y)+(mf011.w * mf011.y)+(mf012.w * mf012.y)+(mf013.w * mf013.y)+(mf014.w * mf014.y)+(mf015.w * mf015.y)+(mf016.w * mf016.y)+(mf017.w * mf017.y)+(mf018.w * mf018.y)+
      (mf019.w * mf019.y)+(mf020.w * mf020.y)+(mf021.w * mf021.y)+(mf022.w * mf022.y)+(mf023.w * mf023.y)+(mf024.w * mf024.y)+(mf025.w * mf025.y)+(mf026.w * mf026.y)+(mf027.w * mf027.y)+
      (mf028.w * mf028.y)+(mf029.w * mf029.y)+(mf030.w * mf030.y)+(mf031.w * mf031.y)+(mf032.w * mf032.y)+(mf033.w * mf033.y)+(mf034.w * mf034.y)+(mf035.w * mf035.y)+(mf036.w * mf036.y)+
      (mf037.w * mf037.y)+(mf038.w * mf038.y)+(mf039.w * mf039.y)+(mf040.w * mf040.y)+(mf041.w * mf041.y)+(mf042.w * mf042.y)+(mf043.w * mf043.y)+(mf044.w * mf044.y)+(mf045.w * mf045.y)+
      (mf046.w * mf046.y)+(mf047.w * mf047.y)+(mf048.w * mf048.y)+(mf049.w * mf049.y)+(mf050.w * mf050.y)+(mf051.w * mf051.y)+(mf052.w * mf052.y)+(mf053.w * mf053.y)+(mf054.w * mf054.y)+
      (mf055.w * mf055.y)+(mf056.w * mf056.y)+(mf057.w * mf057.y)+(mf058.w * mf058.y)+(mf059.w * mf059.y)+(mf060.w * mf060.y)+(mf061.w * mf061.y)+(mf062.w * mf062.y)+(mf063.w * mf063.y)+
      (mf064.w * mf064.y)+(mf065.w * mf065.y)+(mf066.w * mf066.y)+(mf067.w * mf067.y)+(mf068.w * mf068.y)+(mf069.w * mf069.y)+(mf070.w * mf070.y)+(mf071.w * mf071.y)+(mf072.w * mf072.y)+
      (mf073.w * mf073.y)+(mf074.w * mf074.y)+(mf075.w * mf075.y)+(mf076.w * mf076.y)+(mf077.w * mf077.y)+(mf078.w * mf078.y)+(mf079.w * mf079.y)+(mf080.w * mf080.y)+(mf081.w * mf081.y)+
      (mf082.w * mf082.y)+(mf083.w * mf083.y)+(mf084.w * mf084.y)+(mf085.w * mf085.y)+(mf086.w * mf086.y)+(mf087.w * mf087.y)+(mf088.w * mf088.y)+(mf089.w * mf089.y)+(mf090.w * mf090.y)+
      (mf091.w * mf091.y)+(mf092.w * mf092.y)+(mf093.w * mf093.y)+(mf094.w * mf094.y)+(mf095.w * mf095.y)+(mf096.w * mf096.y)+(mf097.w * mf097.y)+(mf098.w * mf098.y)+(mf099.w * mf099.y)+
      (mf100.w * mf100.y)+(mf101.w * mf101.y)+(mf102.w * mf102.y)+(mf103.w * mf103.y)+(mf104.w * mf104.y)+(mf105.w * mf105.y)+(mf106.w * mf106.y)+(mf107.w * mf107.y)+(mf108.w * mf108.y)+
      (mf109.w * mf109.y)+(mf110.w * mf110.y)+(mf111.w * mf111.y)+(mf112.w * mf112.y)+(mf113.w * mf113.y)+(mf114.w * mf114.y)+(mf115.w * mf115.y)+(mf116.w * mf116.y)+(mf117.w * mf117.y)+
      (mf118.w * mf118.y)+(mf119.w * mf119.y)+(mf120.w * mf120.y)+(mf121.w * mf121.y)+(mf122.w * mf122.y)+(mf123.w * mf123.y)+(mf124.w * mf124.y)+(mf125.w * mf125.y)+(mf126.w * mf126.y)+
      (mf127.w * mf127.y);

    ai.z += (mf000.w * mf000.z)+
      (mf001.w * mf001.z)+(mf002.w * mf002.z)+(mf003.w * mf003.z)+(mf004.w * mf004.z)+(mf005.w * mf005.z)+(mf006.w * mf006.z)+(mf007.w * mf007.z)+(mf008.w * mf008.z)+(mf009.w * mf009.z)+
      (mf010.w * mf010.z)+(mf011.w * mf011.z)+(mf012.w * mf012.z)+(mf013.w * mf013.z)+(mf014.w * mf014.z)+(mf015.w * mf015.z)+(mf016.w * mf016.z)+(mf017.w * mf017.z)+(mf018.w * mf018.z)+
      (mf019.w * mf019.z)+(mf020.w * mf020.z)+(mf021.w * mf021.z)+(mf022.w * mf022.z)+(mf023.w * mf023.z)+(mf024.w * mf024.z)+(mf025.w * mf025.z)+(mf026.w * mf026.z)+(mf027.w * mf027.z)+
      (mf028.w * mf028.z)+(mf029.w * mf029.z)+(mf030.w * mf030.z)+(mf031.w * mf031.z)+(mf032.w * mf032.z)+(mf033.w * mf033.z)+(mf034.w * mf034.z)+(mf035.w * mf035.z)+(mf036.w * mf036.z)+
      (mf037.w * mf037.z)+(mf038.w * mf038.z)+(mf039.w * mf039.z)+(mf040.w * mf040.z)+(mf041.w * mf041.z)+(mf042.w * mf042.z)+(mf043.w * mf043.z)+(mf044.w * mf044.z)+(mf045.w * mf045.z)+
      (mf046.w * mf046.z)+(mf047.w * mf047.z)+(mf048.w * mf048.z)+(mf049.w * mf049.z)+(mf050.w * mf050.z)+(mf051.w * mf051.z)+(mf052.w * mf052.z)+(mf053.w * mf053.z)+(mf054.w * mf054.z)+
      (mf055.w * mf055.z)+(mf056.w * mf056.z)+(mf057.w * mf057.z)+(mf058.w * mf058.z)+(mf059.w * mf059.z)+(mf060.w * mf060.z)+(mf061.w * mf061.z)+(mf062.w * mf062.z)+(mf063.w * mf063.z)+
      (mf064.w * mf064.z)+(mf065.w * mf065.z)+(mf066.w * mf066.z)+(mf067.w * mf067.z)+(mf068.w * mf068.z)+(mf069.w * mf069.z)+(mf070.w * mf070.z)+(mf071.w * mf071.z)+(mf072.w * mf072.z)+
      (mf073.w * mf073.z)+(mf074.w * mf074.z)+(mf075.w * mf075.z)+(mf076.w * mf076.z)+(mf077.w * mf077.z)+(mf078.w * mf078.z)+(mf079.w * mf079.z)+(mf080.w * mf080.z)+(mf081.w * mf081.z)+
      (mf082.w * mf082.z)+(mf083.w * mf083.z)+(mf084.w * mf084.z)+(mf085.w * mf085.z)+(mf086.w * mf086.z)+(mf087.w * mf087.z)+(mf088.w * mf088.z)+(mf089.w * mf089.z)+(mf090.w * mf090.z)+
      (mf091.w * mf091.z)+(mf092.w * mf092.z)+(mf093.w * mf093.z)+(mf094.w * mf094.z)+(mf095.w * mf095.z)+(mf096.w * mf096.z)+(mf097.w * mf097.z)+(mf098.w * mf098.z)+(mf099.w * mf099.z)+
      (mf100.w * mf100.z)+(mf101.w * mf101.z)+(mf102.w * mf102.z)+(mf103.w * mf103.z)+(mf104.w * mf104.z)+(mf105.w * mf105.z)+(mf106.w * mf106.z)+(mf107.w * mf107.z)+(mf108.w * mf108.z)+
      (mf109.w * mf109.z)+(mf110.w * mf110.z)+(mf111.w * mf111.z)+(mf112.w * mf112.z)+(mf113.w * mf113.z)+(mf114.w * mf114.z)+(mf115.w * mf115.z)+(mf116.w * mf116.z)+(mf117.w * mf117.z)+
      (mf118.w * mf118.z)+(mf119.w * mf119.z)+(mf120.w * mf120.z)+(mf121.w * mf121.z)+(mf122.w * mf122.z)+(mf123.w * mf123.z)+(mf124.w * mf124.z)+(mf125.w * mf125.z)+(mf126.w * mf126.z)+
      (mf127.w * mf127.z);
  }

  if(i<ni){
    g_fi[i]      = ai.x;
    g_fi[i+ni]   = ai.y;
    g_fi[i+ni*2] = ai.z;
  }
}

#include "api.h"
