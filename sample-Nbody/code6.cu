#include "hip/hip_runtime.h"
//Time-stamp: <2010-06-18 11:31:00 hamada>
// Copyright(C) 2008 by 
// Tsuyoshi Hamada <hamada@progrape.jp>

#include <iostream>
#include "hip/hip_runtime.h"
#include "cutil.h"

namespace libcunbody{

  using namespace std;

  // I want to move these member functions into class cunbody.
  __device__ float4 dev_inter(float4 xi, float4 xj);
  __device__ float4 dev_inter_001(float4 xi, float4 xj);
  __device__ float4 dev_inter_011(float4 xi, float4 xj, float* pot);
  __device__ float4 dev_apot(float4 xi, float4 xj, float4 apot);
  __global__ void cunbody_kernel_tree_001(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_002(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_003(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_011(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_012(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_013(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);
  __global__ void cunbody_kernel_tree_999(float4 *xilist, float4 *xjlist, float4 *apotlist, unsigned int *ioffset, unsigned int *joffset);

  // -------------------------------------- TUNING PROPERTY
#define KERNEL_TYPE (0x002)                 // (XXX) = (Jark tyep, Pot type, Acc type), 0: not implement 
  //#define KERNEL_TYPE (0x013)
#define ACC_TYPE (1)
#define MAX_WALK (256)
#define CUDA_MALLOC_TYPE (0)
#define CUDA_MALLOC_HOST_TYPE (0)
#define DEV_OPEN_STRATEGY (1)
#define NTHRE (128)
  //----------------------------------------.

  class cunbody
  {
  private:
    bool is_open;
    int host_pid;      // host process id
    int host_tid;      // host thread Id
    float4 *dev_xi;
    float4 *dev_xj;
    float4 *dev_apot;
    unsigned int *dev_ioff;
    unsigned int *dev_joff;
    unsigned int isize;
    unsigned int jsize;
    unsigned int max_isize;
    unsigned int max_jsize;

//#define ENABLE_DITAILED_OPENMSG

    void dev_check(){
      int ndev;
      CUDA_SAFE_CALL(hipGetDeviceCount(&ndev));
      if(ndev == 0){
	fprintf(stdout, "ndev = %d @ %s|%d\n", ndev, __FILE__, __LINE__);
	fprintf(stdout, "There is no GPUs.\n");
	exit(-1);
      }else{
	int dev = host_pid % ndev;
	CUDA_SAFE_CALL(hipSetDevice(dev));
	printf("[%d]  : hipSetDevice to %d-th GPU\n",        host_pid, dev+1);
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
	  printf("There is no device supporting CUDA.\n");
	}
#if defined(ENABLE_DITAILED_OPENMSG)
	printf("[%d]  : Major revision number:  %d\n",        host_pid, deviceProp.major);
	printf("[%d]  : Minor revision number:  %d\n",        host_pid, deviceProp.minor);
        printf("[%d]  : core clock rate:  %.2f GHz\n",        host_pid, deviceProp.clockRate * 1e-6f);
#  if  (CUDART_VERSION >= 2000)
	printf("[%d]  : Number of cores:  %d\n",              host_pid, 8 * deviceProp.multiProcessorCount);
        printf("[%d]  : Concurrent copy and execution: %s\n", host_pid, deviceProp.deviceOverlap ? "Yes" : "No");
#  endif
#endif

      }
    }

    void dev_open(int hpid, int htid) {
      //      CUT_DEVICE_INIT(, ); // thread safe ?
      host_pid = hpid;
      host_tid = htid;
      fprintf(stdout, "[%d] CUNBODY-1 library: rev.hamada20080920 (^-^)v\n", host_pid);
#if defined(ENABLE_DITAILED_OPENMSG)
      fprintf(stdout, "[%d]  open GPU by host thread %d\n", host_pid, host_tid);
      fprintf(stdout, "[%d]  : KERNEL_TYPE %03x\n",     host_pid, KERNEL_TYPE);
      fprintf(stdout, "[%d]  : MAX_WALK %d\n",          host_pid, MAX_WALK);
      fprintf(stdout, "[%d]  : NTHRE    %d\n",          host_pid, NTHRE);
      fprintf(stdout, "[%d]  : ACC_TYPE %d\n",          host_pid, ACC_TYPE);
      fprintf(stdout, "[%d]  : CUDA_MALLOC_TYPE %d\n",  host_pid, CUDA_MALLOC_TYPE);
      fprintf(stdout, "[%d]  : CUDA_MALLOC_HOST_TYPE %d\n", host_pid, CUDA_MALLOC_HOST_TYPE);
      fprintf(stdout, "[%d]  : DEV_OPEN_STRATEGY %d\n", host_pid, DEV_OPEN_STRATEGY);
#endif
      dev_check();
      //      max_isize = 1500000;
      //      max_jsize = (this->jsize)*2;

      //      max_isize = (this->isize)*3;
      //      max_jsize = (this->jsize) + (this->jsize)>>1;

      //theta=0.5
      //      max_isize = 380000;
      //      max_jsize = 2100000;

      //theta=0.4
#if 0
      max_isize = 380000;
      max_jsize = 4000000;
#else
      max_isize = 520000;
      max_jsize = 4200000;
#endif

      CUDA_SAFE_CALL(hipMalloc((void **)&dev_xi,   (NTHRE + max_isize) * sizeof(float4)));
      CUDA_SAFE_CALL(hipMalloc((void **)&dev_apot, max_isize * sizeof(float4)));
      CUDA_SAFE_CALL(hipMalloc((void **)&dev_xj,   max_jsize * sizeof(float4)));
      CUDA_SAFE_CALL(hipMalloc((void **)&dev_ioff, (MAX_WALK + 1) * sizeof(unsigned int)));
      CUDA_SAFE_CALL(hipMalloc((void **)&dev_joff, (MAX_WALK + 1) * sizeof(unsigned int)));
      printf("[%d-proc, %d-thread] ******** hipMalloc in total at dev_open() : %d MB (j=%d, i=%d)(j=%d MB, i=%d MB)\n", host_pid, host_tid, 
	     (
	      (NTHRE + max_isize) * sizeof(float4) + 
	      max_isize * sizeof(float4) + 
	      max_jsize * sizeof(float4) + 
	      (MAX_WALK + 1) * sizeof(unsigned int) + 
	      (MAX_WALK + 1) * sizeof(unsigned int)
	      )/(1024*1024), //                           <----------- total(i, j, offset)

	     max_jsize, max_isize,

	     (max_jsize * sizeof(float4))/(1024*1024), // <----------- only j

	     ((NTHRE + max_isize) * sizeof(float4) +
	      max_isize * sizeof(float4))/(1024*1024)  // <----------- only i
	     );

      is_open = true;
    }

    void dev_close(void) {
      CUDA_SAFE_CALL(hipFree(dev_xi));
      CUDA_SAFE_CALL(hipFree(dev_xj));
      CUDA_SAFE_CALL(hipFree(dev_apot));
      CUDA_SAFE_CALL(hipFree(dev_ioff));
      CUDA_SAFE_CALL(hipFree(dev_joff));
    }

  public:

    cunbody() {
      is_open = false;
      host_pid = -1;
      host_tid = -1;
      max_isize = 0;
      max_jsize = 0;
      dev_xi = NULL;
      dev_xj = NULL;
      dev_apot = NULL;
      dev_ioff = NULL;
      dev_joff = NULL;
    }

    ~cunbody() {
      this->dev_close();
      dev_xi = NULL;
      dev_xj = NULL;
      dev_apot = NULL;
      dev_ioff = NULL;
      dev_joff = NULL;
      max_isize = 0;
      max_jsize = 0;
      host_tid = -1;
      host_pid = -1;
      is_open = false;
    }

    void vforce_open(int host_pid, int host_tid)
    {
      if(is_open == false) this->dev_open(host_pid, host_tid);
    }

    void vforce_mp(int host_pid,
		   int host_tid, 
		float4 xilist[], 
		float4 xjlist[], 
		float4 apotlist[], 
		unsigned int ioff[], 
		unsigned int joff[], 
		unsigned int nwalk) 
    {
      isize = ioff[nwalk];
      jsize = joff[nwalk];

      if(is_open == false) this->dev_open(host_pid, host_tid);

#define WARNING_CUDA_MALLOC

      if(isize > max_isize){
	int isize_bak = isize;
	isize = (int)(isize*1.1);
	max_isize = isize_bak;
#if defined(WARNING_CUDA_MALLOC)
	int megabyte = ((NTHRE + isize) * sizeof(float4)+isize * sizeof(float4))>>20;
	printf("[%d @%d]================== hipMalloc for i : %d MB\n", host_pid, host_tid, megabyte);
#endif
	CUDA_SAFE_CALL(hipFree(dev_xi));
	CUDA_SAFE_CALL(hipFree(dev_apot));
	CUDA_SAFE_CALL(hipMalloc((void **)&dev_xi,   (NTHRE + isize) * sizeof(float4)));
	CUDA_SAFE_CALL(hipMalloc((void **)&dev_apot, isize * sizeof(float4)));
	isize = isize_bak;
      }

      if(jsize > max_jsize){
	max_jsize = jsize;
#if defined(WARNING_CUDA_MALLOC)
	int megabyte = (jsize * sizeof(float4))>>20;
	printf("[%d @%d]------------------ hipMalloc for j : %d MB\n",  host_pid, host_tid, megabyte);
#endif
	CUDA_SAFE_CALL(hipFree(dev_xj));
	CUDA_SAFE_CALL(hipMalloc((void **)&dev_xj,   jsize * sizeof(float4)));
      }

      CUDA_SAFE_CALL(hipMemcpy(dev_xi, xilist, isize * sizeof(float4), hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(dev_xj, xjlist, jsize * sizeof(float4), hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(dev_ioff, ioff, (nwalk + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(dev_joff, joff, (nwalk + 1) * sizeof(unsigned int), hipMemcpyHostToDevice));

      dim3 grid(nwalk);
      dim3 threads(NTHRE);

#if   (KERNEL_TYPE == 0x001)
      cunbody_kernel_tree_001 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#elif (KERNEL_TYPE == 0x002)
      cunbody_kernel_tree_002 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#elif (KERNEL_TYPE == 0x011)
      cunbody_kernel_tree_011 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#elif (KERNEL_TYPE == 0x012)
      cunbody_kernel_tree_012 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#elif (KERNEL_TYPE == 0x013)
      cunbody_kernel_tree_013 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#elif (KERNEL_TYPE == 0x003)
      cunbody_kernel_tree_003 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff); // has some bug (# of registers overflow ?)
#elif (KERNEL_TYPE == 0x999)
      cunbody_kernel_tree_999 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff); // has some bug
#else
      cunbody_kernel_tree_000 <<< grid, threads >>> (dev_xi, dev_xj, dev_apot, dev_ioff, dev_joff);
#endif

      CUT_CHECK_ERROR("KERNEL EXECUTION FAILED");
      CUDA_SAFE_CALL(hipMemcpy(apotlist, dev_apot, isize * sizeof(float4), hipMemcpyDeviceToHost));
    }

  }; // class cunbody __END__

}; // namespace libcunbody __END__  ----------------------------------------------------------------------------------



#define MAX_OMP_THRE (4)
static libcunbody::cunbody cunObj[MAX_OMP_THRE];

extern "C"
void vforce(int host_tid, float4 xilist[], float4 xjlist[], float4 apotlist[], unsigned int ioff[], unsigned int joff[], unsigned int nwalk)
{
  using namespace std;
  using namespace libcunbody;
  cunObj[host_tid].vforce_mp(0, host_tid, xilist, xjlist, apotlist, ioff, joff, nwalk);
}

extern "C"
void vforce_mp(int proc_id, int host_tid, float4 xilist[], float4 xjlist[], float4 apotlist[], unsigned int ioff[], unsigned int joff[], unsigned int nwalk)
{
  using namespace std;
  using namespace libcunbody;
  cunObj[host_tid].vforce_mp(proc_id, host_tid, xilist, xjlist, apotlist, ioff, joff, nwalk);
}

extern "C"
void vforce_open(int proc_id, int host_tid)
{
  using namespace std;
  using namespace libcunbody;
  cunObj[host_tid].vforce_open(proc_id, host_tid);
}

#if (NTHRE != 128)
compile-touccha-dame
#endif

namespace libcunbody{

  using namespace std;

  __device__ float4 dev_inter(float4 xi, float4 xj)
  {
    float dx = xj.x - xi.x;
    float dy = xj.y - xi.y;
    float dz = xj.z - xi.z;
    float eps2 = xi.w;
    float mj   = xj.w;

#if (0)
    float r2   = (dz*dz +(dy*dy +(dx*dx+eps2)));
#else
    float r2   = (dx*dx + eps2) + dy*dy + dz*dz;
#endif

    float r1i  = 1/sqrt(r2);
    float r2i  = r1i*r1i;
    float mr3i = mj * r1i * r2i;

#if (0)  
    float4 retval = make_float4(dx, dy, dz, mr3i);
#else
    float4 retval;
    retval.x = dx;
    retval.y = dy;
    retval.z = dz;
    retval.w = mr3i;
#endif

    return retval;
  }

  __global__ 
  void cunbody_kernel_tree_999(float4 *xilist,
				float4 *xjlist,
				float4 *apotlist,
				unsigned int *ioffset,
				unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase+tid];
      float4 apot = make_float4(0,0,0,0);
      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	{
	  float4 dm0 = dev_inter(xi, sj[0]);
	  float4 dm1 = dev_inter(xi, sj[1]);
	  float4 dm2 = dev_inter(xi, sj[2]);
	  float4 dm3 = dev_inter(xi, sj[3]);
	  float4 dm4 = dev_inter(xi, sj[4]);
	  float4 dm5 = dev_inter(xi, sj[5]);
	  float4 dm6 = dev_inter(xi, sj[6]);
	  float4 dm7 = dev_inter(xi, sj[7]);
	  float4 dm8 = dev_inter(xi, sj[8]);
	  float4 dm9 = dev_inter(xi, sj[9]);

	  float4 dm10 = dev_inter(xi, sj[10]);
	  float4 dm11 = dev_inter(xi, sj[11]);
	  float4 dm12 = dev_inter(xi, sj[12]);
	  float4 dm13 = dev_inter(xi, sj[13]);
	  float4 dm14 = dev_inter(xi, sj[14]);
	  float4 dm15 = dev_inter(xi, sj[15]);
	  float4 dm16 = dev_inter(xi, sj[16]);
	  float4 dm17 = dev_inter(xi, sj[17]);
	  float4 dm18 = dev_inter(xi, sj[18]);
	  float4 dm19 = dev_inter(xi, sj[19]);

	  float4 dm20 = dev_inter(xi, sj[20]);
	  float4 dm21 = dev_inter(xi, sj[21]);
	  float4 dm22 = dev_inter(xi, sj[22]);
	  float4 dm23 = dev_inter(xi, sj[23]);
	  float4 dm24 = dev_inter(xi, sj[24]);
	  float4 dm25 = dev_inter(xi, sj[25]);
	  float4 dm26 = dev_inter(xi, sj[26]);
	  float4 dm27 = dev_inter(xi, sj[27]);
	  float4 dm28 = dev_inter(xi, sj[28]);
	  float4 dm29 = dev_inter(xi, sj[29]);

	  float4 dm30 = dev_inter(xi, sj[30]);
	  float4 dm31 = dev_inter(xi, sj[31]);
	  float4 dm32 = dev_inter(xi, sj[32]);
	  float4 dm33 = dev_inter(xi, sj[33]);
	  float4 dm34 = dev_inter(xi, sj[34]);
	  float4 dm35 = dev_inter(xi, sj[35]);
	  float4 dm36 = dev_inter(xi, sj[36]);
	  float4 dm37 = dev_inter(xi, sj[37]);
	  float4 dm38 = dev_inter(xi, sj[38]);
	  float4 dm39 = dev_inter(xi, sj[39]);

	  float4 dm40 = dev_inter(xi, sj[40]);
	  float4 dm41 = dev_inter(xi, sj[41]);
	  float4 dm42 = dev_inter(xi, sj[42]);
	  float4 dm43 = dev_inter(xi, sj[43]);
	  float4 dm44 = dev_inter(xi, sj[44]);
	  float4 dm45 = dev_inter(xi, sj[45]);
	  float4 dm46 = dev_inter(xi, sj[46]);
	  float4 dm47 = dev_inter(xi, sj[47]);
	  float4 dm48 = dev_inter(xi, sj[48]);
	  float4 dm49 = dev_inter(xi, sj[49]);

	  float4 dm50 = dev_inter(xi, sj[50]);
	  float4 dm51 = dev_inter(xi, sj[51]);
	  float4 dm52 = dev_inter(xi, sj[52]);
	  float4 dm53 = dev_inter(xi, sj[53]);
	  float4 dm54 = dev_inter(xi, sj[54]);
	  float4 dm55 = dev_inter(xi, sj[55]);
	  float4 dm56 = dev_inter(xi, sj[56]);
	  float4 dm57 = dev_inter(xi, sj[57]);
	  float4 dm58 = dev_inter(xi, sj[58]);
	  float4 dm59 = dev_inter(xi, sj[59]);

	  float4 dm60 = dev_inter(xi, sj[60]);
	  float4 dm61 = dev_inter(xi, sj[61]);
	  float4 dm62 = dev_inter(xi, sj[62]);
	  float4 dm63 = dev_inter(xi, sj[63]);
	  float4 dm64 = dev_inter(xi, sj[64]);
	  float4 dm65 = dev_inter(xi, sj[65]);
	  float4 dm66 = dev_inter(xi, sj[66]);
	  float4 dm67 = dev_inter(xi, sj[67]);
	  float4 dm68 = dev_inter(xi, sj[68]);
	  float4 dm69 = dev_inter(xi, sj[69]);

	  float4 dm70 = dev_inter(xi, sj[70]);
	  float4 dm71 = dev_inter(xi, sj[71]);
	  float4 dm72 = dev_inter(xi, sj[72]);
	  float4 dm73 = dev_inter(xi, sj[73]);
	  float4 dm74 = dev_inter(xi, sj[74]);
	  float4 dm75 = dev_inter(xi, sj[75]);
	  float4 dm76 = dev_inter(xi, sj[76]);
	  float4 dm77 = dev_inter(xi, sj[77]);
	  float4 dm78 = dev_inter(xi, sj[78]);
	  float4 dm79 = dev_inter(xi, sj[79]);

	  float4 dm80 = dev_inter(xi, sj[80]);
	  float4 dm81 = dev_inter(xi, sj[81]);
	  float4 dm82 = dev_inter(xi, sj[82]);
	  float4 dm83 = dev_inter(xi, sj[83]);
	  float4 dm84 = dev_inter(xi, sj[84]);
	  float4 dm85 = dev_inter(xi, sj[85]);
	  float4 dm86 = dev_inter(xi, sj[86]);
	  float4 dm87 = dev_inter(xi, sj[87]);
	  float4 dm88 = dev_inter(xi, sj[88]);
	  float4 dm89 = dev_inter(xi, sj[89]);

	  float4 dm90 = dev_inter(xi, sj[90]);
	  float4 dm91 = dev_inter(xi, sj[91]);
	  float4 dm92 = dev_inter(xi, sj[92]);
	  float4 dm93 = dev_inter(xi, sj[93]);
	  float4 dm94 = dev_inter(xi, sj[94]);
	  float4 dm95 = dev_inter(xi, sj[95]);
	  float4 dm96 = dev_inter(xi, sj[96]);
	  float4 dm97 = dev_inter(xi, sj[97]);
	  float4 dm98 = dev_inter(xi, sj[98]);
	  float4 dm99 = dev_inter(xi, sj[99]);

	  float4 dm100 = dev_inter(xi, sj[100]);
	  float4 dm101 = dev_inter(xi, sj[101]);
	  float4 dm102 = dev_inter(xi, sj[102]);
	  float4 dm103 = dev_inter(xi, sj[103]);
	  float4 dm104 = dev_inter(xi, sj[104]);
	  float4 dm105 = dev_inter(xi, sj[105]);
	  float4 dm106 = dev_inter(xi, sj[106]);
	  float4 dm107 = dev_inter(xi, sj[107]);
	  float4 dm108 = dev_inter(xi, sj[108]);
	  float4 dm109 = dev_inter(xi, sj[109]);

	  float4 dm110 = dev_inter(xi, sj[110]);
	  float4 dm111 = dev_inter(xi, sj[111]);
	  float4 dm112 = dev_inter(xi, sj[112]);
	  float4 dm113 = dev_inter(xi, sj[113]);
	  float4 dm114 = dev_inter(xi, sj[114]);
	  float4 dm115 = dev_inter(xi, sj[115]);
	  float4 dm116 = dev_inter(xi, sj[116]);
	  float4 dm117 = dev_inter(xi, sj[117]);
	  float4 dm118 = dev_inter(xi, sj[118]);
	  float4 dm119 = dev_inter(xi, sj[119]);

	  float4 dm120 = dev_inter(xi, sj[120]);
	  float4 dm121 = dev_inter(xi, sj[121]);
	  float4 dm122 = dev_inter(xi, sj[122]);
	  float4 dm123 = dev_inter(xi, sj[123]);
	  float4 dm124 = dev_inter(xi, sj[124]);
	  float4 dm125 = dev_inter(xi, sj[125]);
	  float4 dm126 = dev_inter(xi, sj[126]);
	  float4 dm127 = dev_inter(xi, sj[127]);

	  apot.x = dm0.x * dm0.w + dm1.x * dm1.w + dm2.x * dm2.w + dm3.x * dm3.w + dm4.x * dm4.w + dm5.x * dm5.w + dm6.x * dm6.w + dm7.x * dm7.w + dm8.x * dm8.w + dm9.x * dm9.w + 
	    dm10.x * dm10.w + dm11.x * dm11.w + dm12.x * dm12.w + dm13.x * dm13.w + dm14.x * dm14.w + dm15.x * dm15.w + dm16.x * dm16.w + dm17.x * dm17.w + dm18.x * dm18.w + dm19.x * dm19.w + 
	    dm20.x * dm20.w + dm21.x * dm21.w + dm22.x * dm22.w + dm23.x * dm23.w + dm24.x * dm24.w + dm25.x * dm25.w + dm26.x * dm26.w + dm27.x * dm27.w + dm28.x * dm28.w + dm29.x * dm29.w + 
	    dm30.x * dm30.w + dm31.x * dm31.w + dm32.x * dm32.w + dm33.x * dm33.w + dm34.x * dm34.w + dm35.x * dm35.w + dm36.x * dm36.w + dm37.x * dm37.w + dm38.x * dm38.w + dm39.x * dm39.w + 
	    dm40.x * dm40.w + dm41.x * dm41.w + dm42.x * dm42.w + dm43.x * dm43.w + dm44.x * dm44.w + dm45.x * dm45.w + dm46.x * dm46.w + dm47.x * dm47.w + dm48.x * dm48.w + dm49.x * dm49.w + 
	    dm50.x * dm50.w + dm51.x * dm51.w + dm52.x * dm52.w + dm53.x * dm53.w + dm54.x * dm54.w + dm55.x * dm55.w + dm56.x * dm56.w + dm57.x * dm57.w + dm58.x * dm58.w + dm59.x * dm59.w + 
	    dm60.x * dm60.w + dm61.x * dm61.w + dm62.x * dm62.w + dm63.x * dm63.w + dm64.x * dm64.w + dm65.x * dm65.w + dm66.x * dm66.w + dm67.x * dm67.w + dm68.x * dm68.w + dm69.x * dm69.w + 
	    dm70.x * dm70.w + dm71.x * dm71.w + dm72.x * dm72.w + dm73.x * dm73.w + dm74.x * dm74.w + dm75.x * dm75.w + dm76.x * dm76.w + dm77.x * dm77.w + dm78.x * dm78.w + dm79.x * dm79.w + 
	    dm80.x * dm80.w + dm81.x * dm81.w + dm82.x * dm82.w + dm83.x * dm83.w + dm84.x * dm84.w + dm85.x * dm85.w + dm86.x * dm86.w + dm87.x * dm87.w + dm88.x * dm88.w + dm89.x * dm89.w + 
	    dm90.x * dm90.w + dm91.x * dm91.w + dm92.x * dm92.w + dm93.x * dm93.w + dm94.x * dm94.w + dm95.x * dm95.w + dm96.x * dm96.w + dm97.x * dm97.w + dm98.x * dm98.w + dm99.x * dm99.w + 
	    dm100.x * dm100.w + dm101.x * dm101.w + dm102.x * dm102.w + dm103.x * dm103.w + dm104.x * dm104.w + dm105.x * dm105.w + dm106.x * dm106.w + dm107.x * dm107.w + dm108.x * dm108.w + dm109.x * dm109.w + 
	    dm110.x * dm110.w + dm111.x * dm111.w + dm112.x * dm112.w + dm113.x * dm113.w + dm114.x * dm114.w + dm115.x * dm115.w + dm116.x * dm116.w + dm117.x * dm117.w + dm118.x * dm118.w + dm119.x * dm119.w + 
	    dm120.x * dm120.w + dm121.x * dm121.w + dm122.x * dm122.w + dm123.x * dm123.w + dm124.x * dm124.w + dm125.x * dm125.w + dm126.x * dm126.w + dm127.x * dm127.w;

	  apot.y = dm0.y * dm0.w + dm1.y * dm1.w + dm2.y * dm2.w + dm3.y * dm3.w + dm4.y * dm4.w + dm5.y * dm5.w + dm6.y * dm6.w + dm7.y * dm7.w + dm8.y * dm8.w + dm9.y * dm9.w + 
	    dm10.y * dm10.w + dm11.y * dm11.w + dm12.y * dm12.w + dm13.y * dm13.w + dm14.y * dm14.w + dm15.y * dm15.w + dm16.y * dm16.w + dm17.y * dm17.w + dm18.y * dm18.w + dm19.y * dm19.w + 
	    dm20.y * dm20.w + dm21.y * dm21.w + dm22.y * dm22.w + dm23.y * dm23.w + dm24.y * dm24.w + dm25.y * dm25.w + dm26.y * dm26.w + dm27.y * dm27.w + dm28.y * dm28.w + dm29.y * dm29.w + 
	    dm30.y * dm30.w + dm31.y * dm31.w + dm32.y * dm32.w + dm33.y * dm33.w + dm34.y * dm34.w + dm35.y * dm35.w + dm36.y * dm36.w + dm37.y * dm37.w + dm38.y * dm38.w + dm39.y * dm39.w + 
	    dm40.y * dm40.w + dm41.y * dm41.w + dm42.y * dm42.w + dm43.y * dm43.w + dm44.y * dm44.w + dm45.y * dm45.w + dm46.y * dm46.w + dm47.y * dm47.w + dm48.y * dm48.w + dm49.y * dm49.w + 
	    dm50.y * dm50.w + dm51.y * dm51.w + dm52.y * dm52.w + dm53.y * dm53.w + dm54.y * dm54.w + dm55.y * dm55.w + dm56.y * dm56.w + dm57.y * dm57.w + dm58.y * dm58.w + dm59.y * dm59.w + 
	    dm60.y * dm60.w + dm61.y * dm61.w + dm62.y * dm62.w + dm63.y * dm63.w + dm64.y * dm64.w + dm65.y * dm65.w + dm66.y * dm66.w + dm67.y * dm67.w + dm68.y * dm68.w + dm69.y * dm69.w + 
	    dm70.y * dm70.w + dm71.y * dm71.w + dm72.y * dm72.w + dm73.y * dm73.w + dm74.y * dm74.w + dm75.y * dm75.w + dm76.y * dm76.w + dm77.y * dm77.w + dm78.y * dm78.w + dm79.y * dm79.w + 
	    dm80.y * dm80.w + dm81.y * dm81.w + dm82.y * dm82.w + dm83.y * dm83.w + dm84.y * dm84.w + dm85.y * dm85.w + dm86.y * dm86.w + dm87.y * dm87.w + dm88.y * dm88.w + dm89.y * dm89.w + 
	    dm90.y * dm90.w + dm91.y * dm91.w + dm92.y * dm92.w + dm93.y * dm93.w + dm94.y * dm94.w + dm95.y * dm95.w + dm96.y * dm96.w + dm97.y * dm97.w + dm98.y * dm98.w + dm99.y * dm99.w + 
	    dm100.y * dm100.w + dm101.y * dm101.w + dm102.y * dm102.w + dm103.y * dm103.w + dm104.y * dm104.w + dm105.y * dm105.w + dm106.y * dm106.w + dm107.y * dm107.w + dm108.y * dm108.w + dm109.y * dm109.w + 
	    dm110.y * dm110.w + dm111.y * dm111.w + dm112.y * dm112.w + dm113.y * dm113.w + dm114.y * dm114.w + dm115.y * dm115.w + dm116.y * dm116.w + dm117.y * dm117.w + dm118.y * dm118.w + dm119.y * dm119.w + 
	    dm120.y * dm120.w + dm121.y * dm121.w + dm122.y * dm122.w + dm123.y * dm123.w + dm124.y * dm124.w + dm125.y * dm125.w + dm126.y * dm126.w + dm127.y * dm127.w;

	  apot.z = dm0.z * dm0.w + dm1.z * dm1.w + dm2.z * dm2.w + dm3.z * dm3.w + dm4.z * dm4.w + dm5.z * dm5.w + dm6.z * dm6.w + dm7.z * dm7.w + dm8.z * dm8.w + dm9.z * dm9.w + 
	    dm10.z * dm10.w + dm11.z * dm11.w + dm12.z * dm12.w + dm13.z * dm13.w + dm14.z * dm14.w + dm15.z * dm15.w + dm16.z * dm16.w + dm17.z * dm17.w + dm18.z * dm18.w + dm19.z * dm19.w + 
	    dm20.z * dm20.w + dm21.z * dm21.w + dm22.z * dm22.w + dm23.z * dm23.w + dm24.z * dm24.w + dm25.z * dm25.w + dm26.z * dm26.w + dm27.z * dm27.w + dm28.z * dm28.w + dm29.z * dm29.w + 
	    dm30.z * dm30.w + dm31.z * dm31.w + dm32.z * dm32.w + dm33.z * dm33.w + dm34.z * dm34.w + dm35.z * dm35.w + dm36.z * dm36.w + dm37.z * dm37.w + dm38.z * dm38.w + dm39.z * dm39.w + 
	    dm40.z * dm40.w + dm41.z * dm41.w + dm42.z * dm42.w + dm43.z * dm43.w + dm44.z * dm44.w + dm45.z * dm45.w + dm46.z * dm46.w + dm47.z * dm47.w + dm48.z * dm48.w + dm49.z * dm49.w + 
	    dm50.z * dm50.w + dm51.z * dm51.w + dm52.z * dm52.w + dm53.z * dm53.w + dm54.z * dm54.w + dm55.z * dm55.w + dm56.z * dm56.w + dm57.z * dm57.w + dm58.z * dm58.w + dm59.z * dm59.w + 
	    dm60.z * dm60.w + dm61.z * dm61.w + dm62.z * dm62.w + dm63.z * dm63.w + dm64.z * dm64.w + dm65.z * dm65.w + dm66.z * dm66.w + dm67.z * dm67.w + dm68.z * dm68.w + dm69.z * dm69.w + 
	    dm70.z * dm70.w + dm71.z * dm71.w + dm72.z * dm72.w + dm73.z * dm73.w + dm74.z * dm74.w + dm75.z * dm75.w + dm76.z * dm76.w + dm77.z * dm77.w + dm78.z * dm78.w + dm79.z * dm79.w + 
	    dm80.z * dm80.w + dm81.z * dm81.w + dm82.z * dm82.w + dm83.z * dm83.w + dm84.z * dm84.w + dm85.z * dm85.w + dm86.z * dm86.w + dm87.z * dm87.w + dm88.z * dm88.w + dm89.z * dm89.w + 
	    dm90.z * dm90.w + dm91.z * dm91.w + dm92.z * dm92.w + dm93.z * dm93.w + dm94.z * dm94.w + dm95.z * dm95.w + dm96.z * dm96.w + dm97.z * dm97.w + dm98.z * dm98.w + dm99.z * dm99.w + 
	    dm100.z * dm100.w + dm101.z * dm101.w + dm102.z * dm102.w + dm103.z * dm103.w + dm104.z * dm104.w + dm105.z * dm105.w + dm106.z * dm106.w + dm107.z * dm107.w + dm108.z * dm108.w + dm109.z * dm109.w + 
	    dm110.z * dm110.w + dm111.z * dm111.w + dm112.z * dm112.w + dm113.z * dm113.w + dm114.z * dm114.w + dm115.z * dm115.w + dm116.z * dm116.w + dm117.z * dm117.w + dm118.z * dm118.w + dm119.z * dm119.w + 
	    dm120.z * dm120.w + dm121.z * dm121.w + dm122.z * dm122.w + dm123.z * dm123.w + dm124.z * dm124.w + dm125.z * dm125.w + dm126.z * dm126.w + dm127.z * dm127.w;

	}
      }
      if(ibase+tid < iend){
	apotlist[ibase+tid] = apot;
      }
    }
  }



  __device__ 
  float4 dev_inter_001(float4 xi, float4 xj)
  {
    float dx = xj.x - xi.x;
    float dy = xj.y - xi.y;
    float dz = xj.z - xi.z;
    float eps2 = xi.w;
    float mj   = xj.w;
#if (0)
    float r2 = dx*dz + (dy*dy + (dx*dx + eps2));
#else
    float r2 = (dx*dx + eps2) + dy*dy + dz*dz;
#endif
    //	if(r2 == eps2) return (a,p)

    float r1i = 1/sqrt(r2);
    //  float r1i = rsqrt(r2);

    float r2i = r1i*r1i;
    float mr3i = mj * r1i * r2i;
    float4 retval;

    retval.x = dx; 
    retval.y = dy; 
    retval.z = dz;
    retval.w = mr3i; 

    return (retval);
  }



  __global__ 
  void cunbody_kernel_tree_001(float4 *xilist,
				float4 *xjlist,
				float4 *apotlist,
				unsigned int *ioffset,
				unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0,0,0,0);
      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[0]);
	  dm1 = dev_inter_001(xi, sj[1]);
	  dm2 = dev_inter_001(xi, sj[2]);
	  dm3 = dev_inter_001(xi, sj[3]);
	  dm4 = dev_inter_001(xi, sj[4]);
	  dm5 = dev_inter_001(xi, sj[5]);
	  dm6 = dev_inter_001(xi, sj[6]);
	  dm7 = dev_inter_001(xi, sj[7]);
	  dm8 = dev_inter_001(xi, sj[8]);
	  dm9 = dev_inter_001(xi, sj[9]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[10]);
	  dm1 = dev_inter_001(xi, sj[11]);
	  dm2 = dev_inter_001(xi, sj[12]);
	  dm3 = dev_inter_001(xi, sj[13]);
	  dm4 = dev_inter_001(xi, sj[14]);
	  dm5 = dev_inter_001(xi, sj[15]);
	  dm6 = dev_inter_001(xi, sj[16]);
	  dm7 = dev_inter_001(xi, sj[17]);
	  dm8 = dev_inter_001(xi, sj[18]);
	  dm9 = dev_inter_001(xi, sj[19]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[20]);
	  dm1 = dev_inter_001(xi, sj[21]);
	  dm2 = dev_inter_001(xi, sj[22]);
	  dm3 = dev_inter_001(xi, sj[23]);
	  dm4 = dev_inter_001(xi, sj[24]);
	  dm5 = dev_inter_001(xi, sj[25]);
	  dm6 = dev_inter_001(xi, sj[26]);
	  dm7 = dev_inter_001(xi, sj[27]);
	  dm8 = dev_inter_001(xi, sj[28]);
	  dm9 = dev_inter_001(xi, sj[29]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[30]);
	  dm1 = dev_inter_001(xi, sj[31]);
	  dm2 = dev_inter_001(xi, sj[32]);
	  dm3 = dev_inter_001(xi, sj[33]);
	  dm4 = dev_inter_001(xi, sj[34]);
	  dm5 = dev_inter_001(xi, sj[35]);
	  dm6 = dev_inter_001(xi, sj[36]);
	  dm7 = dev_inter_001(xi, sj[37]);
	  dm8 = dev_inter_001(xi, sj[38]);
	  dm9 = dev_inter_001(xi, sj[39]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[40]);
	  dm1 = dev_inter_001(xi, sj[41]);
	  dm2 = dev_inter_001(xi, sj[42]);
	  dm3 = dev_inter_001(xi, sj[43]);
	  dm4 = dev_inter_001(xi, sj[44]);
	  dm5 = dev_inter_001(xi, sj[45]);
	  dm6 = dev_inter_001(xi, sj[46]);
	  dm7 = dev_inter_001(xi, sj[47]);
	  dm8 = dev_inter_001(xi, sj[48]);
	  dm9 = dev_inter_001(xi, sj[49]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[50]);
	  dm1 = dev_inter_001(xi, sj[51]);
	  dm2 = dev_inter_001(xi, sj[52]);
	  dm3 = dev_inter_001(xi, sj[53]);
	  dm4 = dev_inter_001(xi, sj[54]);
	  dm5 = dev_inter_001(xi, sj[55]);
	  dm6 = dev_inter_001(xi, sj[56]);
	  dm7 = dev_inter_001(xi, sj[57]);
	  dm8 = dev_inter_001(xi, sj[58]);
	  dm9 = dev_inter_001(xi, sj[59]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[60]);
	  dm1 = dev_inter_001(xi, sj[61]);
	  dm2 = dev_inter_001(xi, sj[62]);
	  dm3 = dev_inter_001(xi, sj[63]);
	  dm4 = dev_inter_001(xi, sj[64]);
	  dm5 = dev_inter_001(xi, sj[65]);
	  dm6 = dev_inter_001(xi, sj[66]);
	  dm7 = dev_inter_001(xi, sj[67]);
	  dm8 = dev_inter_001(xi, sj[68]);
	  dm9 = dev_inter_001(xi, sj[69]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[70]);
	  dm1 = dev_inter_001(xi, sj[71]);
	  dm2 = dev_inter_001(xi, sj[72]);
	  dm3 = dev_inter_001(xi, sj[73]);
	  dm4 = dev_inter_001(xi, sj[74]);
	  dm5 = dev_inter_001(xi, sj[75]);
	  dm6 = dev_inter_001(xi, sj[76]);
	  dm7 = dev_inter_001(xi, sj[77]);
	  dm8 = dev_inter_001(xi, sj[78]);
	  dm9 = dev_inter_001(xi, sj[79]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[80]);
	  dm1 = dev_inter_001(xi, sj[81]);
	  dm2 = dev_inter_001(xi, sj[82]);
	  dm3 = dev_inter_001(xi, sj[83]);
	  dm4 = dev_inter_001(xi, sj[84]);
	  dm5 = dev_inter_001(xi, sj[85]);
	  dm6 = dev_inter_001(xi, sj[86]);
	  dm7 = dev_inter_001(xi, sj[87]);
	  dm8 = dev_inter_001(xi, sj[88]);
	  dm9 = dev_inter_001(xi, sj[89]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[90]);
	  dm1 = dev_inter_001(xi, sj[91]);
	  dm2 = dev_inter_001(xi, sj[92]);
	  dm3 = dev_inter_001(xi, sj[93]);
	  dm4 = dev_inter_001(xi, sj[94]);
	  dm5 = dev_inter_001(xi, sj[95]);
	  dm6 = dev_inter_001(xi, sj[96]);
	  dm7 = dev_inter_001(xi, sj[97]);
	  dm8 = dev_inter_001(xi, sj[98]);
	  dm9 = dev_inter_001(xi, sj[99]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[100]);
	  dm1 = dev_inter_001(xi, sj[101]);
	  dm2 = dev_inter_001(xi, sj[102]);
	  dm3 = dev_inter_001(xi, sj[103]);
	  dm4 = dev_inter_001(xi, sj[104]);
	  dm5 = dev_inter_001(xi, sj[105]);
	  dm6 = dev_inter_001(xi, sj[106]);
	  dm7 = dev_inter_001(xi, sj[107]);
	  dm8 = dev_inter_001(xi, sj[108]);
	  dm9 = dev_inter_001(xi, sj[109]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[110]);
	  dm1 = dev_inter_001(xi, sj[111]);
	  dm2 = dev_inter_001(xi, sj[112]);
	  dm3 = dev_inter_001(xi, sj[113]);
	  dm4 = dev_inter_001(xi, sj[114]);
	  dm5 = dev_inter_001(xi, sj[115]);
	  dm6 = dev_inter_001(xi, sj[116]);
	  dm7 = dev_inter_001(xi, sj[117]);
	  dm8 = dev_inter_001(xi, sj[118]);
	  dm9 = dev_inter_001(xi, sj[119]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7;//, dm8, dm9;
	  dm0 = dev_inter_001(xi, sj[120]);
	  dm1 = dev_inter_001(xi, sj[121]);
	  dm2 = dev_inter_001(xi, sj[122]);
	  dm3 = dev_inter_001(xi, sj[123]);
	  dm4 = dev_inter_001(xi, sj[124]);
	  dm5 = dev_inter_001(xi, sj[125]);
	  dm6 = dev_inter_001(xi, sj[126]);
	  dm7 = dev_inter_001(xi, sj[127]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w);// + (dm8.x*dm8.w) + (dm9.x*dm9.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w);// + (dm8.y*dm8.w) + (dm9.y*dm9.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w);// + (dm8.z*dm8.w) + (dm9.z*dm9.w);
	}
      }
      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
    }
  }


  __global__ 
  void cunbody_kernel_tree_002(float4 *xilist,
				float4 *xjlist,
				float4 *apotlist,
				unsigned int *ioffset,
				unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0,0,0,0);
      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	__syncthreads(); // bugfixed 2008/09/19
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[0]);
	  dm1 = dev_inter_001(xi, sj[1]);
	  dm2 = dev_inter_001(xi, sj[2]);
	  dm3 = dev_inter_001(xi, sj[3]);
	  dm4 = dev_inter_001(xi, sj[4]);
	  dm5 = dev_inter_001(xi, sj[5]);
	  dm6 = dev_inter_001(xi, sj[6]);
	  dm7 = dev_inter_001(xi, sj[7]);
	  dm8 = dev_inter_001(xi, sj[8]);
	  dm9 = dev_inter_001(xi, sj[9]);
	  dm10 = dev_inter_001(xi, sj[10]);
	  dm11 = dev_inter_001(xi, sj[11]);
	  dm12 = dev_inter_001(xi, sj[12]);
	  dm13 = dev_inter_001(xi, sj[13]);
	  dm14 = dev_inter_001(xi, sj[14]);
	  dm15 = dev_inter_001(xi, sj[15]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[16]);
	  dm1 = dev_inter_001(xi, sj[17]);
	  dm2 = dev_inter_001(xi, sj[18]);
	  dm3 = dev_inter_001(xi, sj[19]);
	  dm4 = dev_inter_001(xi, sj[20]);
	  dm5 = dev_inter_001(xi, sj[21]);
	  dm6 = dev_inter_001(xi, sj[22]);
	  dm7 = dev_inter_001(xi, sj[23]);
	  dm8 = dev_inter_001(xi, sj[24]);
	  dm9 = dev_inter_001(xi, sj[25]);
	  dm10 = dev_inter_001(xi, sj[26]);
	  dm11 = dev_inter_001(xi, sj[27]);
	  dm12 = dev_inter_001(xi, sj[28]);
	  dm13 = dev_inter_001(xi, sj[29]);
	  dm14 = dev_inter_001(xi, sj[30]);
	  dm15 = dev_inter_001(xi, sj[31]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[32]);
	  dm1 = dev_inter_001(xi, sj[33]);
	  dm2 = dev_inter_001(xi, sj[34]);
	  dm3 = dev_inter_001(xi, sj[35]);
	  dm4 = dev_inter_001(xi, sj[36]);
	  dm5 = dev_inter_001(xi, sj[37]);
	  dm6 = dev_inter_001(xi, sj[38]);
	  dm7 = dev_inter_001(xi, sj[39]);
	  dm8 = dev_inter_001(xi, sj[40]);
	  dm9 = dev_inter_001(xi, sj[41]);
	  dm10 = dev_inter_001(xi, sj[42]);
	  dm11 = dev_inter_001(xi, sj[43]);
	  dm12 = dev_inter_001(xi, sj[44]);
	  dm13 = dev_inter_001(xi, sj[45]);
	  dm14 = dev_inter_001(xi, sj[46]);
	  dm15 = dev_inter_001(xi, sj[47]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[48]);
	  dm1 = dev_inter_001(xi, sj[49]);
	  dm2 = dev_inter_001(xi, sj[50]);
	  dm3 = dev_inter_001(xi, sj[51]);
	  dm4 = dev_inter_001(xi, sj[52]);
	  dm5 = dev_inter_001(xi, sj[53]);
	  dm6 = dev_inter_001(xi, sj[54]);
	  dm7 = dev_inter_001(xi, sj[55]);
	  dm8 = dev_inter_001(xi, sj[56]);
	  dm9 = dev_inter_001(xi, sj[57]);
	  dm10 = dev_inter_001(xi, sj[58]);
	  dm11 = dev_inter_001(xi, sj[59]);
	  dm12 = dev_inter_001(xi, sj[60]);
	  dm13 = dev_inter_001(xi, sj[61]);
	  dm14 = dev_inter_001(xi, sj[62]);
	  dm15 = dev_inter_001(xi, sj[63]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[64]);
	  dm1 = dev_inter_001(xi, sj[65]);
	  dm2 = dev_inter_001(xi, sj[66]);
	  dm3 = dev_inter_001(xi, sj[67]);
	  dm4 = dev_inter_001(xi, sj[68]);
	  dm5 = dev_inter_001(xi, sj[69]);
	  dm6 = dev_inter_001(xi, sj[70]);
	  dm7 = dev_inter_001(xi, sj[71]);
	  dm8 = dev_inter_001(xi, sj[72]);
	  dm9 = dev_inter_001(xi, sj[73]);
	  dm10 = dev_inter_001(xi, sj[74]);
	  dm11 = dev_inter_001(xi, sj[75]);
	  dm12 = dev_inter_001(xi, sj[76]);
	  dm13 = dev_inter_001(xi, sj[77]);
	  dm14 = dev_inter_001(xi, sj[78]);
	  dm15 = dev_inter_001(xi, sj[79]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[80]);
	  dm1 = dev_inter_001(xi, sj[81]);
	  dm2 = dev_inter_001(xi, sj[82]);
	  dm3 = dev_inter_001(xi, sj[83]);
	  dm4 = dev_inter_001(xi, sj[84]);
	  dm5 = dev_inter_001(xi, sj[85]);
	  dm6 = dev_inter_001(xi, sj[86]);
	  dm7 = dev_inter_001(xi, sj[87]);
	  dm8 = dev_inter_001(xi, sj[88]);
	  dm9 = dev_inter_001(xi, sj[89]);
	  dm10 = dev_inter_001(xi, sj[90]);
	  dm11 = dev_inter_001(xi, sj[91]);
	  dm12 = dev_inter_001(xi, sj[92]);
	  dm13 = dev_inter_001(xi, sj[93]);
	  dm14 = dev_inter_001(xi, sj[94]);
	  dm15 = dev_inter_001(xi, sj[95]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[96]);
	  dm1 = dev_inter_001(xi, sj[97]);
	  dm2 = dev_inter_001(xi, sj[98]);
	  dm3 = dev_inter_001(xi, sj[99]);
	  dm4 = dev_inter_001(xi, sj[100]);
	  dm5 = dev_inter_001(xi, sj[101]);
	  dm6 = dev_inter_001(xi, sj[102]);
	  dm7 = dev_inter_001(xi, sj[103]);
	  dm8 = dev_inter_001(xi, sj[104]);
	  dm9 = dev_inter_001(xi, sj[105]);
	  dm10 = dev_inter_001(xi, sj[106]);
	  dm11 = dev_inter_001(xi, sj[107]);
	  dm12 = dev_inter_001(xi, sj[108]);
	  dm13 = dev_inter_001(xi, sj[109]);
	  dm14 = dev_inter_001(xi, sj[110]);
	  dm15 = dev_inter_001(xi, sj[111]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_001(xi, sj[112]);
	  dm1 = dev_inter_001(xi, sj[113]);
	  dm2 = dev_inter_001(xi, sj[114]);
	  dm3 = dev_inter_001(xi, sj[115]);
	  dm4 = dev_inter_001(xi, sj[116]);
	  dm5 = dev_inter_001(xi, sj[117]);
	  dm6 = dev_inter_001(xi, sj[118]);
	  dm7 = dev_inter_001(xi, sj[119]);
	  dm8 = dev_inter_001(xi, sj[120]);
	  dm9 = dev_inter_001(xi, sj[121]);
	  dm10 = dev_inter_001(xi, sj[122]);
	  dm11 = dev_inter_001(xi, sj[123]);
	  dm12 = dev_inter_001(xi, sj[124]);
	  dm13 = dev_inter_001(xi, sj[125]);
	  dm14 = dev_inter_001(xi, sj[126]);
	  dm15 = dev_inter_001(xi, sj[127]);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

      }
      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
    }
  }

  __global__ 
  void cunbody_kernel_tree_003(float4 *xilist,
				float4 *xjlist,
				float4 *apotlist,
				unsigned int *ioffset,
				unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0,0,0,0);
      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	{
	  float4 dm0 = dev_inter_001(xi, sj[0]);
	  float4 dm1 = dev_inter_001(xi, sj[1]);
	  float4 dm2 = dev_inter_001(xi, sj[2]);
	  float4 dm3 = dev_inter_001(xi, sj[3]);
	  float4 dm4 = dev_inter_001(xi, sj[4]);
	  float4 dm5 = dev_inter_001(xi, sj[5]);
	  float4 dm6 = dev_inter_001(xi, sj[6]);
	  float4 dm7 = dev_inter_001(xi, sj[7]);
	  float4 dm8 = dev_inter_001(xi, sj[8]);
	  float4 dm9 = dev_inter_001(xi, sj[9]);

	  float4 dm10 = dev_inter_001(xi, sj[0]);
	  float4 dm11 = dev_inter_001(xi, sj[1]);
	  float4 dm12 = dev_inter_001(xi, sj[2]);
	  float4 dm13 = dev_inter_001(xi, sj[3]);
	  float4 dm14 = dev_inter_001(xi, sj[4]);
	  float4 dm15 = dev_inter_001(xi, sj[5]);
	  float4 dm16 = dev_inter_001(xi, sj[6]);
	  float4 dm17 = dev_inter_001(xi, sj[7]);
	  float4 dm18 = dev_inter_001(xi, sj[8]);
	  float4 dm19 = dev_inter_001(xi, sj[9]);

	  float4 dm20 = dev_inter_001(xi, sj[0]);
	  float4 dm21 = dev_inter_001(xi, sj[1]);
	  float4 dm22 = dev_inter_001(xi, sj[2]);
	  float4 dm23 = dev_inter_001(xi, sj[3]);
	  float4 dm24 = dev_inter_001(xi, sj[4]);
	  float4 dm25 = dev_inter_001(xi, sj[5]);
	  float4 dm26 = dev_inter_001(xi, sj[6]);
	  float4 dm27 = dev_inter_001(xi, sj[7]);
	  float4 dm28 = dev_inter_001(xi, sj[8]);
	  float4 dm29 = dev_inter_001(xi, sj[9]);

	  float4 dm30 = dev_inter_001(xi, sj[0]);
	  float4 dm31 = dev_inter_001(xi, sj[1]);
	  float4 dm32 = dev_inter_001(xi, sj[2]);
	  float4 dm33 = dev_inter_001(xi, sj[3]);
	  float4 dm34 = dev_inter_001(xi, sj[4]);
	  float4 dm35 = dev_inter_001(xi, sj[5]);
	  float4 dm36 = dev_inter_001(xi, sj[6]);
	  float4 dm37 = dev_inter_001(xi, sj[7]);
	  float4 dm38 = dev_inter_001(xi, sj[8]);
	  float4 dm39 = dev_inter_001(xi, sj[9]);

	  float4 dm40 = dev_inter_001(xi, sj[0]);
	  float4 dm41 = dev_inter_001(xi, sj[1]);
	  float4 dm42 = dev_inter_001(xi, sj[2]);
	  float4 dm43 = dev_inter_001(xi, sj[3]);
	  float4 dm44 = dev_inter_001(xi, sj[4]);
	  float4 dm45 = dev_inter_001(xi, sj[5]);
	  float4 dm46 = dev_inter_001(xi, sj[6]);
	  float4 dm47 = dev_inter_001(xi, sj[7]);
	  float4 dm48 = dev_inter_001(xi, sj[8]);
	  float4 dm49 = dev_inter_001(xi, sj[9]);

	  float4 dm50 = dev_inter_001(xi, sj[0]);
	  float4 dm51 = dev_inter_001(xi, sj[1]);
	  float4 dm52 = dev_inter_001(xi, sj[2]);
	  float4 dm53 = dev_inter_001(xi, sj[3]);
	  float4 dm54 = dev_inter_001(xi, sj[4]);
	  float4 dm55 = dev_inter_001(xi, sj[5]);
	  float4 dm56 = dev_inter_001(xi, sj[6]);
	  float4 dm57 = dev_inter_001(xi, sj[7]);
	  float4 dm58 = dev_inter_001(xi, sj[8]);
	  float4 dm59 = dev_inter_001(xi, sj[9]);

	  float4 dm60 = dev_inter_001(xi, sj[0]);
	  float4 dm61 = dev_inter_001(xi, sj[1]);
	  float4 dm62 = dev_inter_001(xi, sj[2]);
	  float4 dm63 = dev_inter_001(xi, sj[3]);
	  float4 dm64 = dev_inter_001(xi, sj[4]);
	  float4 dm65 = dev_inter_001(xi, sj[5]);
	  float4 dm66 = dev_inter_001(xi, sj[6]);
	  float4 dm67 = dev_inter_001(xi, sj[7]);
	  float4 dm68 = dev_inter_001(xi, sj[8]);
	  float4 dm69 = dev_inter_001(xi, sj[9]);

	  float4 dm70 = dev_inter_001(xi, sj[0]);
	  float4 dm71 = dev_inter_001(xi, sj[1]);
	  float4 dm72 = dev_inter_001(xi, sj[2]);
	  float4 dm73 = dev_inter_001(xi, sj[3]);
	  float4 dm74 = dev_inter_001(xi, sj[4]);
	  float4 dm75 = dev_inter_001(xi, sj[5]);
	  float4 dm76 = dev_inter_001(xi, sj[6]);
	  float4 dm77 = dev_inter_001(xi, sj[7]);
	  float4 dm78 = dev_inter_001(xi, sj[8]);
	  float4 dm79 = dev_inter_001(xi, sj[9]);

	  float4 dm80 = dev_inter_001(xi, sj[0]);
	  float4 dm81 = dev_inter_001(xi, sj[1]);
	  float4 dm82 = dev_inter_001(xi, sj[2]);
	  float4 dm83 = dev_inter_001(xi, sj[3]);
	  float4 dm84 = dev_inter_001(xi, sj[4]);
	  float4 dm85 = dev_inter_001(xi, sj[5]);
	  float4 dm86 = dev_inter_001(xi, sj[6]);
	  float4 dm87 = dev_inter_001(xi, sj[7]);
	  float4 dm88 = dev_inter_001(xi, sj[8]);
	  float4 dm89 = dev_inter_001(xi, sj[9]);

	  float4 dm90 = dev_inter_001(xi, sj[0]);
	  float4 dm91 = dev_inter_001(xi, sj[1]);
	  float4 dm92 = dev_inter_001(xi, sj[2]);
	  float4 dm93 = dev_inter_001(xi, sj[3]);
	  float4 dm94 = dev_inter_001(xi, sj[4]);
	  float4 dm95 = dev_inter_001(xi, sj[5]);
	  float4 dm96 = dev_inter_001(xi, sj[6]);
	  float4 dm97 = dev_inter_001(xi, sj[7]);
	  float4 dm98 = dev_inter_001(xi, sj[8]);
	  float4 dm99 = dev_inter_001(xi, sj[9]);

	  float4 dm100 = dev_inter_001(xi, sj[0]);
	  float4 dm101 = dev_inter_001(xi, sj[1]);
	  float4 dm102 = dev_inter_001(xi, sj[2]);
	  float4 dm103 = dev_inter_001(xi, sj[3]);
	  float4 dm104 = dev_inter_001(xi, sj[4]);
	  float4 dm105 = dev_inter_001(xi, sj[5]);
	  float4 dm106 = dev_inter_001(xi, sj[6]);
	  float4 dm107 = dev_inter_001(xi, sj[7]);
	  float4 dm108 = dev_inter_001(xi, sj[8]);
	  float4 dm109 = dev_inter_001(xi, sj[9]);

	  float4 dm110 = dev_inter_001(xi, sj[0]);
	  float4 dm111 = dev_inter_001(xi, sj[1]);
	  float4 dm112 = dev_inter_001(xi, sj[2]);
	  float4 dm113 = dev_inter_001(xi, sj[3]);
	  float4 dm114 = dev_inter_001(xi, sj[4]);
	  float4 dm115 = dev_inter_001(xi, sj[5]);
	  float4 dm116 = dev_inter_001(xi, sj[6]);
	  float4 dm117 = dev_inter_001(xi, sj[7]);
	  float4 dm118 = dev_inter_001(xi, sj[8]);
	  float4 dm119 = dev_inter_001(xi, sj[9]);

	  float4 dm120 = dev_inter_001(xi, sj[0]);
	  float4 dm121 = dev_inter_001(xi, sj[1]);
	  float4 dm122 = dev_inter_001(xi, sj[2]);
	  float4 dm123 = dev_inter_001(xi, sj[3]);
	  float4 dm124 = dev_inter_001(xi, sj[4]);
	  float4 dm125 = dev_inter_001(xi, sj[5]);
	  float4 dm126 = dev_inter_001(xi, sj[6]);
	  float4 dm127 = dev_inter_001(xi, sj[7]);


#if (ACC_TYPE==1)
	  apot.x += \
	    (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) + (dm8.x*dm8.w) + (dm9.x*dm9.w) + \
	    (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w) + (dm16.x*dm16.w) + (dm17.x*dm17.w) + (dm18.x*dm18.w) + (dm19.x*dm19.w) + \
	    (dm20.x*dm20.w) + (dm21.x*dm21.w) + (dm22.x*dm22.w) + (dm23.x*dm23.w) + (dm24.x*dm24.w) + (dm25.x*dm25.w) + (dm26.x*dm26.w) + (dm27.x*dm27.w) + (dm28.x*dm28.w) + (dm29.x*dm29.w) + \
	    (dm30.x*dm30.w) + (dm31.x*dm31.w) + (dm32.x*dm32.w) + (dm33.x*dm33.w) + (dm34.x*dm34.w) + (dm35.x*dm35.w) + (dm36.x*dm36.w) + (dm37.x*dm37.w) + (dm38.x*dm38.w) + (dm39.x*dm39.w) + \
	    (dm40.x*dm40.w) + (dm41.x*dm41.w) + (dm42.x*dm42.w) + (dm43.x*dm43.w) + (dm44.x*dm44.w) + (dm45.x*dm45.w) + (dm46.x*dm46.w) + (dm47.x*dm47.w) + (dm48.x*dm48.w) + (dm49.x*dm49.w) + \
	    (dm50.x*dm50.w) + (dm51.x*dm51.w) + (dm52.x*dm52.w) + (dm53.x*dm53.w) + (dm54.x*dm54.w) + (dm55.x*dm55.w) + (dm56.x*dm56.w) + (dm57.x*dm57.w) + (dm58.x*dm58.w) + (dm59.x*dm59.w) + \
	    (dm60.x*dm60.w) + (dm61.x*dm61.w) + (dm62.x*dm62.w) + (dm63.x*dm63.w) + (dm64.x*dm64.w) + (dm65.x*dm65.w) + (dm66.x*dm66.w) + (dm67.x*dm67.w) + (dm68.x*dm68.w) + (dm69.x*dm69.w) + \
	    (dm70.x*dm70.w) + (dm71.x*dm71.w) + (dm72.x*dm72.w) + (dm73.x*dm73.w) + (dm74.x*dm74.w) + (dm75.x*dm75.w) + (dm76.x*dm76.w) + (dm77.x*dm77.w) + (dm78.x*dm78.w) + (dm79.x*dm79.w) + \
	    (dm80.x*dm80.w) + (dm81.x*dm81.w) + (dm82.x*dm82.w) + (dm83.x*dm83.w) + (dm84.x*dm84.w) + (dm85.x*dm85.w) + (dm86.x*dm86.w) + (dm87.x*dm87.w) + (dm88.x*dm88.w) + (dm89.x*dm89.w) + \
	    (dm90.x*dm90.w) + (dm91.x*dm91.w) + (dm92.x*dm92.w) + (dm93.x*dm93.w) + (dm94.x*dm94.w) + (dm95.x*dm95.w) + (dm96.x*dm96.w) + (dm97.x*dm97.w) + (dm98.x*dm98.w) + (dm99.x*dm99.w) + \
	    (dm100.x*dm100.w) + (dm101.x*dm101.w) + (dm102.x*dm102.w) + (dm103.x*dm103.w) + (dm104.x*dm104.w) + (dm105.x*dm105.w) + (dm106.x*dm106.w) + (dm107.x*dm107.w) + (dm108.x*dm108.w) + (dm109.x*dm109.w) + \
	    (dm110.x*dm110.w) + (dm111.x*dm111.w) + (dm112.x*dm112.w) + (dm113.x*dm113.w) + (dm114.x*dm114.w) + (dm115.x*dm115.w) + (dm116.x*dm116.w) + (dm117.x*dm117.w) + (dm118.x*dm118.w) + (dm119.x*dm119.w) + \
	    (dm120.x*dm120.w) + (dm121.x*dm121.w) + (dm122.x*dm122.w) + (dm123.x*dm123.w) + (dm124.x*dm124.w) + (dm125.x*dm125.w) + (dm126.x*dm126.w) + (dm127.x*dm127.w);

	  apot.y += \
	    (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) + (dm8.y*dm8.w) + (dm9.y*dm9.w) + \
	    (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w) + (dm16.y*dm16.w) + (dm17.y*dm17.w) + (dm18.y*dm18.w) + (dm19.y*dm19.w) + \
	    (dm20.y*dm20.w) + (dm21.y*dm21.w) + (dm22.y*dm22.w) + (dm23.y*dm23.w) + (dm24.y*dm24.w) + (dm25.y*dm25.w) + (dm26.y*dm26.w) + (dm27.y*dm27.w) + (dm28.y*dm28.w) + (dm29.y*dm29.w) + \
	    (dm30.y*dm30.w) + (dm31.y*dm31.w) + (dm32.y*dm32.w) + (dm33.y*dm33.w) + (dm34.y*dm34.w) + (dm35.y*dm35.w) + (dm36.y*dm36.w) + (dm37.y*dm37.w) + (dm38.y*dm38.w) + (dm39.y*dm39.w) + \
	    (dm40.y*dm40.w) + (dm41.y*dm41.w) + (dm42.y*dm42.w) + (dm43.y*dm43.w) + (dm44.y*dm44.w) + (dm45.y*dm45.w) + (dm46.y*dm46.w) + (dm47.y*dm47.w) + (dm48.y*dm48.w) + (dm49.y*dm49.w) + \
	    (dm50.y*dm50.w) + (dm51.y*dm51.w) + (dm52.y*dm52.w) + (dm53.y*dm53.w) + (dm54.y*dm54.w) + (dm55.y*dm55.w) + (dm56.y*dm56.w) + (dm57.y*dm57.w) + (dm58.y*dm58.w) + (dm59.y*dm59.w) + \
	    (dm60.y*dm60.w) + (dm61.y*dm61.w) + (dm62.y*dm62.w) + (dm63.y*dm63.w) + (dm64.y*dm64.w) + (dm65.y*dm65.w) + (dm66.y*dm66.w) + (dm67.y*dm67.w) + (dm68.y*dm68.w) + (dm69.y*dm69.w) + \
	    (dm70.y*dm70.w) + (dm71.y*dm71.w) + (dm72.y*dm72.w) + (dm73.y*dm73.w) + (dm74.y*dm74.w) + (dm75.y*dm75.w) + (dm76.y*dm76.w) + (dm77.y*dm77.w) + (dm78.y*dm78.w) + (dm79.y*dm79.w) + \
	    (dm80.y*dm80.w) + (dm81.y*dm81.w) + (dm82.y*dm82.w) + (dm83.y*dm83.w) + (dm84.y*dm84.w) + (dm85.y*dm85.w) + (dm86.y*dm86.w) + (dm87.y*dm87.w) + (dm88.y*dm88.w) + (dm89.y*dm89.w) + \
	    (dm90.y*dm90.w) + (dm91.y*dm91.w) + (dm92.y*dm92.w) + (dm93.y*dm93.w) + (dm94.y*dm94.w) + (dm95.y*dm95.w) + (dm96.y*dm96.w) + (dm97.y*dm97.w) + (dm98.y*dm98.w) + (dm99.y*dm99.w) + \
	    (dm100.y*dm100.w) + (dm101.y*dm101.w) + (dm102.y*dm102.w) + (dm103.y*dm103.w) + (dm104.y*dm104.w) + (dm105.y*dm105.w) + (dm106.y*dm106.w) + (dm107.y*dm107.w) + (dm108.y*dm108.w) + (dm109.y*dm109.w) + \
	    (dm110.y*dm110.w) + (dm111.y*dm111.w) + (dm112.y*dm112.w) + (dm113.y*dm113.w) + (dm114.y*dm114.w) + (dm115.y*dm115.w) + (dm116.y*dm116.w) + (dm117.y*dm117.w) + (dm118.y*dm118.w) + (dm119.y*dm119.w) + \
	    (dm120.y*dm120.w) + (dm121.y*dm121.w) + (dm122.y*dm122.w) + (dm123.y*dm123.w) + (dm124.y*dm124.w) + (dm125.y*dm125.w) + (dm126.y*dm126.w) + (dm127.y*dm127.w);

	  apot.z += \
	    (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) + (dm8.z*dm8.w) + (dm9.z*dm9.w) + \
	    (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w) + (dm16.z*dm16.w) + (dm17.z*dm17.w) + (dm18.z*dm18.w) + (dm19.z*dm19.w) + \
	    (dm20.z*dm20.w) + (dm21.z*dm21.w) + (dm22.z*dm22.w) + (dm23.z*dm23.w) + (dm24.z*dm24.w) + (dm25.z*dm25.w) + (dm26.z*dm26.w) + (dm27.z*dm27.w) + (dm28.z*dm28.w) + (dm29.z*dm29.w) + \
	    (dm30.z*dm30.w) + (dm31.z*dm31.w) + (dm32.z*dm32.w) + (dm33.z*dm33.w) + (dm34.z*dm34.w) + (dm35.z*dm35.w) + (dm36.z*dm36.w) + (dm37.z*dm37.w) + (dm38.z*dm38.w) + (dm39.z*dm39.w) + \
	    (dm40.z*dm40.w) + (dm41.z*dm41.w) + (dm42.z*dm42.w) + (dm43.z*dm43.w) + (dm44.z*dm44.w) + (dm45.z*dm45.w) + (dm46.z*dm46.w) + (dm47.z*dm47.w) + (dm48.z*dm48.w) + (dm49.z*dm49.w) + \
	    (dm50.z*dm50.w) + (dm51.z*dm51.w) + (dm52.z*dm52.w) + (dm53.z*dm53.w) + (dm54.z*dm54.w) + (dm55.z*dm55.w) + (dm56.z*dm56.w) + (dm57.z*dm57.w) + (dm58.z*dm58.w) + (dm59.z*dm59.w) + \
	    (dm60.z*dm60.w) + (dm61.z*dm61.w) + (dm62.z*dm62.w) + (dm63.z*dm63.w) + (dm64.z*dm64.w) + (dm65.z*dm65.w) + (dm66.z*dm66.w) + (dm67.z*dm67.w) + (dm68.z*dm68.w) + (dm69.z*dm69.w) + \
	    (dm70.z*dm70.w) + (dm71.z*dm71.w) + (dm72.z*dm72.w) + (dm73.z*dm73.w) + (dm74.z*dm74.w) + (dm75.z*dm75.w) + (dm76.z*dm76.w) + (dm77.z*dm77.w) + (dm78.z*dm78.w) + (dm79.z*dm79.w) + \
	    (dm80.z*dm80.w) + (dm81.z*dm81.w) + (dm82.z*dm82.w) + (dm83.z*dm83.w) + (dm84.z*dm84.w) + (dm85.z*dm85.w) + (dm86.z*dm86.w) + (dm87.z*dm87.w) + (dm88.z*dm88.w) + (dm89.z*dm89.w) + \
	    (dm90.z*dm90.w) + (dm91.z*dm91.w) + (dm92.z*dm92.w) + (dm93.z*dm93.w) + (dm94.z*dm94.w) + (dm95.z*dm95.w) + (dm96.z*dm96.w) + (dm97.z*dm97.w) + (dm98.z*dm98.w) + (dm99.z*dm99.w) + \
	    (dm100.z*dm100.w) + (dm101.z*dm101.w) + (dm102.z*dm102.w) + (dm103.z*dm103.w) + (dm104.z*dm104.w) + (dm105.z*dm105.w) + (dm106.z*dm106.w) + (dm107.z*dm107.w) + (dm108.z*dm108.w) + (dm109.z*dm109.w) + \
	    (dm110.z*dm110.w) + (dm111.z*dm111.w) + (dm112.z*dm112.w) + (dm113.z*dm113.w) + (dm114.z*dm114.w) + (dm115.z*dm115.w) + (dm116.z*dm116.w) + (dm117.z*dm117.w) + (dm118.z*dm118.w) + (dm119.z*dm119.w) + \
	    (dm120.z*dm120.w) + (dm121.z*dm121.w) + (dm122.z*dm122.w) + (dm123.z*dm123.w) + (dm124.z*dm124.w) + (dm125.z*dm125.w) + (dm126.z*dm126.w) + (dm127.z*dm127.w);

#else 
	  apot.x += \
	    (dm0.x*dm0.w + dm1.x*dm1.w + dm2.x*dm2.w + dm3.x*dm3.w + dm4.x*dm4.w + dm5.x*dm5.w + dm6.x*dm6.w + dm7.x*dm7.w + dm8.x*dm8.w + dm9.x*dm9.w + \
	     dm10.x*dm10.w + dm11.x*dm11.w + dm12.x*dm12.w + dm13.x*dm13.w + dm14.x*dm14.w + dm15.x*dm15.w + dm16.x*dm16.w + dm17.x*dm17.w + dm18.x*dm18.w + dm19.x*dm19.w + \
	     dm20.x*dm20.w + dm21.x*dm21.w + dm22.x*dm22.w + dm23.x*dm23.w + dm24.x*dm24.w + dm25.x*dm25.w + dm26.x*dm26.w + dm27.x*dm27.w + dm28.x*dm28.w + dm29.x*dm29.w + \
	     dm30.x*dm30.w + dm31.x*dm31.w + dm32.x*dm32.w + dm33.x*dm33.w + dm34.x*dm34.w + dm35.x*dm35.w + dm36.x*dm36.w + dm37.x*dm37.w + dm38.x*dm38.w + dm39.x*dm39.w + \
	     dm40.x*dm40.w + dm41.x*dm41.w + dm42.x*dm42.w + dm43.x*dm43.w + dm44.x*dm44.w + dm45.x*dm45.w + dm46.x*dm46.w + dm47.x*dm47.w + dm48.x*dm48.w + dm49.x*dm49.w + \
	     dm50.x*dm50.w + dm51.x*dm51.w + dm52.x*dm52.w + dm53.x*dm53.w + dm54.x*dm54.w + dm55.x*dm55.w + dm56.x*dm56.w + dm57.x*dm57.w + dm58.x*dm58.w + dm59.x*dm59.w + \
	     dm60.x*dm60.w + dm61.x*dm61.w + dm62.x*dm62.w + dm63.x*dm63.w) +\
	    (dm64.x*dm64.w + dm65.x*dm65.w + dm66.x*dm66.w + dm67.x*dm67.w + dm68.x*dm68.w + dm69.x*dm69.w + \
	     dm70.x*dm70.w + dm71.x*dm71.w + dm72.x*dm72.w + dm73.x*dm73.w + dm74.x*dm74.w + dm75.x*dm75.w + dm76.x*dm76.w + dm77.x*dm77.w + dm78.x*dm78.w + dm79.x*dm79.w + \
	     dm80.x*dm80.w + dm81.x*dm81.w + dm82.x*dm82.w + dm83.x*dm83.w + dm84.x*dm84.w + dm85.x*dm85.w + dm86.x*dm86.w + dm87.x*dm87.w + dm88.x*dm88.w + dm89.x*dm89.w + \
	     dm90.x*dm90.w + dm91.x*dm91.w + dm92.x*dm92.w + dm93.x*dm93.w + dm94.x*dm94.w + dm95.x*dm95.w + dm96.x*dm96.w + dm97.x*dm97.w + dm98.x*dm98.w + dm99.x*dm99.w + \
	     dm100.x*dm100.w + dm101.x*dm101.w + dm102.x*dm102.w + dm103.x*dm103.w + dm104.x*dm104.w + dm105.x*dm105.w + dm106.x*dm106.w + dm107.x*dm107.w + dm108.x*dm108.w + dm109.x*dm109.w + \
	     dm110.x*dm110.w + dm111.x*dm111.w + dm112.x*dm112.w + dm113.x*dm113.w + dm114.x*dm114.w + dm115.x*dm115.w + dm116.x*dm116.w + dm117.x*dm117.w + dm118.x*dm118.w + dm119.x*dm119.w + \
	     dm120.x*dm120.w + dm121.x*dm121.w + dm122.x*dm122.w + dm123.x*dm123.w + dm124.x*dm124.w + dm125.x*dm125.w + dm126.x*dm126.w + dm127.x*dm127.w);

	  apot.y += \
	    (dm0.y*dm0.w + dm1.y*dm1.w + dm2.y*dm2.w + dm3.y*dm3.w + dm4.y*dm4.w + dm5.y*dm5.w + dm6.y*dm6.w + dm7.y*dm7.w + dm8.y*dm8.w + dm9.y*dm9.w + \
	     dm10.y*dm10.w + dm11.y*dm11.w + dm12.y*dm12.w + dm13.y*dm13.w + dm14.y*dm14.w + dm15.y*dm15.w + dm16.y*dm16.w + dm17.y*dm17.w + dm18.y*dm18.w + dm19.y*dm19.w + \
	     dm20.y*dm20.w + dm21.y*dm21.w + dm22.y*dm22.w + dm23.y*dm23.w + dm24.y*dm24.w + dm25.y*dm25.w + dm26.y*dm26.w + dm27.y*dm27.w + dm28.y*dm28.w + dm29.y*dm29.w + \
	     dm30.y*dm30.w + dm31.y*dm31.w + dm32.y*dm32.w + dm33.y*dm33.w + dm34.y*dm34.w + dm35.y*dm35.w + dm36.y*dm36.w + dm37.y*dm37.w + dm38.y*dm38.w + dm39.y*dm39.w + \
	     dm40.y*dm40.w + dm41.y*dm41.w + dm42.y*dm42.w + dm43.y*dm43.w + dm44.y*dm44.w + dm45.y*dm45.w + dm46.y*dm46.w + dm47.y*dm47.w + dm48.y*dm48.w + dm49.y*dm49.w + \
	     dm50.y*dm50.w + dm51.y*dm51.w + dm52.y*dm52.w + dm53.y*dm53.w + dm54.y*dm54.w + dm55.y*dm55.w + dm56.y*dm56.w + dm57.y*dm57.w + dm58.y*dm58.w + dm59.y*dm59.w + \
	     dm60.y*dm60.w + dm61.y*dm61.w + dm62.y*dm62.w + dm63.y*dm63.w) +\
	    (dm64.x*dm64.w + dm65.x*dm65.w + dm66.x*dm66.w + dm67.x*dm67.w + dm68.x*dm68.w + dm69.x*dm69.w + \
	     dm70.y*dm70.w + dm71.y*dm71.w + dm72.y*dm72.w + dm73.y*dm73.w + dm74.y*dm74.w + dm75.y*dm75.w + dm76.y*dm76.w + dm77.y*dm77.w + dm78.y*dm78.w + dm79.y*dm79.w + \
	     dm80.y*dm80.w + dm81.y*dm81.w + dm82.y*dm82.w + dm83.y*dm83.w + dm84.y*dm84.w + dm85.y*dm85.w + dm86.y*dm86.w + dm87.y*dm87.w + dm88.y*dm88.w + dm89.y*dm89.w + \
	     dm90.y*dm90.w + dm91.y*dm91.w + dm92.y*dm92.w + dm93.y*dm93.w + dm94.y*dm94.w + dm95.y*dm95.w + dm96.y*dm96.w + dm97.y*dm97.w + dm98.y*dm98.w + dm99.y*dm99.w + \
	     dm100.y*dm100.w + dm101.y*dm101.w + dm102.y*dm102.w + dm103.y*dm103.w + dm104.y*dm104.w + dm105.y*dm105.w + dm106.y*dm106.w + dm107.y*dm107.w + dm108.y*dm108.w + dm109.y*dm109.w + \
	     dm110.y*dm110.w + dm111.y*dm111.w + dm112.y*dm112.w + dm113.y*dm113.w + dm114.y*dm114.w + dm115.y*dm115.w + dm116.y*dm116.w + dm117.y*dm117.w + dm118.y*dm118.w + dm119.y*dm119.w + \
	     dm120.y*dm120.w + dm121.y*dm121.w + dm122.y*dm122.w + dm123.y*dm123.w + dm124.y*dm124.w + dm125.y*dm125.w + dm126.y*dm126.w + dm127.y*dm127.w);

	  apot.z += \
	    (dm0.z*dm0.w + dm1.z*dm1.w + dm2.z*dm2.w + dm3.z*dm3.w + dm4.z*dm4.w + dm5.z*dm5.w + dm6.z*dm6.w + dm7.z*dm7.w + dm8.z*dm8.w + dm9.z*dm9.w + \
	     dm10.z*dm10.w + dm11.z*dm11.w + dm12.z*dm12.w + dm13.z*dm13.w + dm14.z*dm14.w + dm15.z*dm15.w + dm16.z*dm16.w + dm17.z*dm17.w + dm18.z*dm18.w + dm19.z*dm19.w + \
	     dm20.z*dm20.w + dm21.z*dm21.w + dm22.z*dm22.w + dm23.z*dm23.w + dm24.z*dm24.w + dm25.z*dm25.w + dm26.z*dm26.w + dm27.z*dm27.w + dm28.z*dm28.w + dm29.z*dm29.w + \
	     dm30.z*dm30.w + dm31.z*dm31.w + dm32.z*dm32.w + dm33.z*dm33.w + dm34.z*dm34.w + dm35.z*dm35.w + dm36.z*dm36.w + dm37.z*dm37.w + dm38.z*dm38.w + dm39.z*dm39.w + \
	     dm40.z*dm40.w + dm41.z*dm41.w + dm42.z*dm42.w + dm43.z*dm43.w + dm44.z*dm44.w + dm45.z*dm45.w + dm46.z*dm46.w + dm47.z*dm47.w + dm48.z*dm48.w + dm49.z*dm49.w + \
	     dm50.z*dm50.w + dm51.z*dm51.w + dm52.z*dm52.w + dm53.z*dm53.w + dm54.z*dm54.w + dm55.z*dm55.w + dm56.z*dm56.w + dm57.z*dm57.w + dm58.z*dm58.w + dm59.z*dm59.w + \
	     dm60.z*dm60.w + dm61.z*dm61.w + dm62.z*dm62.w + dm63.z*dm63.w) +\
	    (dm64.z*dm64.w + dm65.z*dm65.w + dm66.z*dm66.w + dm67.z*dm67.w + dm68.z*dm68.w + dm69.z*dm69.w + \
	     dm70.z*dm70.w + dm71.z*dm71.w + dm72.z*dm72.w + dm73.z*dm73.w + dm74.z*dm74.w + dm75.z*dm75.w + dm76.z*dm76.w + dm77.z*dm77.w + dm78.z*dm78.w + dm79.z*dm79.w + \
	     dm80.z*dm80.w + dm81.z*dm81.w + dm82.z*dm82.w + dm83.z*dm83.w + dm84.z*dm84.w + dm85.z*dm85.w + dm86.z*dm86.w + dm87.z*dm87.w + dm88.z*dm88.w + dm89.z*dm89.w + \
	     dm90.z*dm90.w + dm91.z*dm91.w + dm92.z*dm92.w + dm93.z*dm93.w + dm94.z*dm94.w + dm95.z*dm95.w + dm96.z*dm96.w + dm97.z*dm97.w + dm98.z*dm98.w + dm99.z*dm99.w + \
	     dm100.z*dm100.w + dm101.z*dm101.w + dm102.z*dm102.w + dm103.z*dm103.w + dm104.z*dm104.w + dm105.z*dm105.w + dm106.z*dm106.w + dm107.z*dm107.w + dm108.z*dm108.w + dm109.z*dm109.w + \
	     dm110.z*dm110.w + dm111.z*dm111.w + dm112.z*dm112.w + dm113.z*dm113.w + dm114.z*dm114.w + dm115.z*dm115.w + dm116.z*dm116.w + dm117.z*dm117.w + dm118.z*dm118.w + dm119.z*dm119.w + \
	     dm120.z*dm120.w + dm121.z*dm121.w + dm122.z*dm122.w + dm123.z*dm123.w + dm124.z*dm124.w + dm125.z*dm125.w + dm126.z*dm126.w + dm127.z*dm127.w);
#endif

	}
      }
      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
    }
  }



  __device__
  float4 dev_inter_011(float4 xi, float4 xj, float* pot)
  {
    float dx = xj.x - xi.x;
    float dy = xj.y - xi.y;
    float dz = xj.z - xi.z;
    float eps2 = xi.w;
    float mj   = xj.w;
    float r2 = (dx*dx + eps2) + dy*dy + dz*dz;
    float r1i  = 1/sqrt(r2);

    if(r2 == eps2) mj = 0.0;
    float r2i = r1i*r1i;
    float mr3i = mj * r1i * r2i;
    float4 retval;

    retval.x = dx; 
    retval.y = dy; 
    retval.z = dz;
    retval.w = mr3i; 
    (*pot) -= mj * r1i;

    return (retval);
  }


  __global__ 
  void cunbody_kernel_tree_011(float4 *xilist,
				float4 *xjlist,
				float4 *apotlist,
				unsigned int *ioffset,
				unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0,0,0,0);
      float pot0 = 0.0;
      float pot1 = 0.0;
      float pot2 = 0.0;
      float pot3 = 0.0;
      float pot4 = 0.0;
      float pot5 = 0.0;
      float pot6 = 0.0;
      float pot7 = 0.0;

      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[0], &pot0);
	  dm1 = dev_inter_011(xi, sj[1], &pot0);
	  dm2 = dev_inter_011(xi, sj[2], &pot0);
	  dm3 = dev_inter_011(xi, sj[3], &pot0);
	  dm4 = dev_inter_011(xi, sj[4], &pot0);
	  dm5 = dev_inter_011(xi, sj[5], &pot0);
	  dm6 = dev_inter_011(xi, sj[6], &pot0);
	  dm7 = dev_inter_011(xi, sj[7], &pot0);
	  dm8 = dev_inter_011(xi, sj[8], &pot0);
	  dm9 = dev_inter_011(xi, sj[9], &pot0);
	  dm10 = dev_inter_011(xi, sj[10], &pot0);
	  dm11 = dev_inter_011(xi, sj[11], &pot0);
	  dm12 = dev_inter_011(xi, sj[12], &pot0);
	  dm13 = dev_inter_011(xi, sj[13], &pot0);
	  dm14 = dev_inter_011(xi, sj[14], &pot0);
	  dm15 = dev_inter_011(xi, sj[15], &pot0);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[16], &pot1);
	  dm1 = dev_inter_011(xi, sj[17], &pot1);
	  dm2 = dev_inter_011(xi, sj[18], &pot1);
	  dm3 = dev_inter_011(xi, sj[19], &pot1);
	  dm4 = dev_inter_011(xi, sj[20], &pot1);
	  dm5 = dev_inter_011(xi, sj[21], &pot1);
	  dm6 = dev_inter_011(xi, sj[22], &pot1);
	  dm7 = dev_inter_011(xi, sj[23], &pot1);
	  dm8 = dev_inter_011(xi, sj[24], &pot1);
	  dm9 = dev_inter_011(xi, sj[25], &pot1);
	  dm10 = dev_inter_011(xi, sj[26], &pot1);
	  dm11 = dev_inter_011(xi, sj[27], &pot1);
	  dm12 = dev_inter_011(xi, sj[28], &pot1);
	  dm13 = dev_inter_011(xi, sj[29], &pot1);
	  dm14 = dev_inter_011(xi, sj[30], &pot1);
	  dm15 = dev_inter_011(xi, sj[31], &pot1);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[32], &pot2);
	  dm1 = dev_inter_011(xi, sj[33], &pot2);
	  dm2 = dev_inter_011(xi, sj[34], &pot2);
	  dm3 = dev_inter_011(xi, sj[35], &pot2);
	  dm4 = dev_inter_011(xi, sj[36], &pot2);
	  dm5 = dev_inter_011(xi, sj[37], &pot2);
	  dm6 = dev_inter_011(xi, sj[38], &pot2);
	  dm7 = dev_inter_011(xi, sj[39], &pot2);
	  dm8 = dev_inter_011(xi, sj[40], &pot2);
	  dm9 = dev_inter_011(xi, sj[41], &pot2);
	  dm10 = dev_inter_011(xi, sj[42], &pot2);
	  dm11 = dev_inter_011(xi, sj[43], &pot2);
	  dm12 = dev_inter_011(xi, sj[44], &pot2);
	  dm13 = dev_inter_011(xi, sj[45], &pot2);
	  dm14 = dev_inter_011(xi, sj[46], &pot2);
	  dm15 = dev_inter_011(xi, sj[47], &pot2);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[48], &pot3);
	  dm1 = dev_inter_011(xi, sj[49], &pot3);
	  dm2 = dev_inter_011(xi, sj[50], &pot3);
	  dm3 = dev_inter_011(xi, sj[51], &pot3);
	  dm4 = dev_inter_011(xi, sj[52], &pot3);
	  dm5 = dev_inter_011(xi, sj[53], &pot3);
	  dm6 = dev_inter_011(xi, sj[54], &pot3);
	  dm7 = dev_inter_011(xi, sj[55], &pot3);
	  dm8 = dev_inter_011(xi, sj[56], &pot3);
	  dm9 = dev_inter_011(xi, sj[57], &pot3);
	  dm10 = dev_inter_011(xi, sj[58], &pot3);
	  dm11 = dev_inter_011(xi, sj[59], &pot3);
	  dm12 = dev_inter_011(xi, sj[60], &pot3);
	  dm13 = dev_inter_011(xi, sj[61], &pot3);
	  dm14 = dev_inter_011(xi, sj[62], &pot3);
	  dm15 = dev_inter_011(xi, sj[63], &pot3);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[64], &pot4);
	  dm1 = dev_inter_011(xi, sj[65], &pot4);
	  dm2 = dev_inter_011(xi, sj[66], &pot4);
	  dm3 = dev_inter_011(xi, sj[67], &pot4);
	  dm4 = dev_inter_011(xi, sj[68], &pot4);
	  dm5 = dev_inter_011(xi, sj[69], &pot4);
	  dm6 = dev_inter_011(xi, sj[70], &pot4);
	  dm7 = dev_inter_011(xi, sj[71], &pot4);
	  dm8 = dev_inter_011(xi, sj[72], &pot4);
	  dm9 = dev_inter_011(xi, sj[73], &pot4);
	  dm10 = dev_inter_011(xi, sj[74], &pot4);
	  dm11 = dev_inter_011(xi, sj[75], &pot4);
	  dm12 = dev_inter_011(xi, sj[76], &pot4);
	  dm13 = dev_inter_011(xi, sj[77], &pot4);
	  dm14 = dev_inter_011(xi, sj[78], &pot4);
	  dm15 = dev_inter_011(xi, sj[79], &pot4);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[80], &pot5);
	  dm1 = dev_inter_011(xi, sj[81], &pot5);
	  dm2 = dev_inter_011(xi, sj[82], &pot5);
	  dm3 = dev_inter_011(xi, sj[83], &pot5);
	  dm4 = dev_inter_011(xi, sj[84], &pot5);
	  dm5 = dev_inter_011(xi, sj[85], &pot5);
	  dm6 = dev_inter_011(xi, sj[86], &pot5);
	  dm7 = dev_inter_011(xi, sj[87], &pot5);
	  dm8 = dev_inter_011(xi, sj[88], &pot5);
	  dm9 = dev_inter_011(xi, sj[89], &pot5);
	  dm10 = dev_inter_011(xi, sj[90], &pot5);
	  dm11 = dev_inter_011(xi, sj[91], &pot5);
	  dm12 = dev_inter_011(xi, sj[92], &pot5);
	  dm13 = dev_inter_011(xi, sj[93], &pot5);
	  dm14 = dev_inter_011(xi, sj[94], &pot5);
	  dm15 = dev_inter_011(xi, sj[95], &pot5);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[96], &pot6);
	  dm1 = dev_inter_011(xi, sj[97], &pot6);
	  dm2 = dev_inter_011(xi, sj[98], &pot6);
	  dm3 = dev_inter_011(xi, sj[99], &pot6);
	  dm4 = dev_inter_011(xi, sj[100], &pot6);
	  dm5 = dev_inter_011(xi, sj[101], &pot6);
	  dm6 = dev_inter_011(xi, sj[102], &pot6);
	  dm7 = dev_inter_011(xi, sj[103], &pot6);
	  dm8 = dev_inter_011(xi, sj[104], &pot6);
	  dm9 = dev_inter_011(xi, sj[105], &pot6);
	  dm10 = dev_inter_011(xi, sj[106], &pot6);
	  dm11 = dev_inter_011(xi, sj[107], &pot6);
	  dm12 = dev_inter_011(xi, sj[108], &pot6);
	  dm13 = dev_inter_011(xi, sj[109], &pot6);
	  dm14 = dev_inter_011(xi, sj[110], &pot6);
	  dm15 = dev_inter_011(xi, sj[111], &pot6);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

	{
	  float4 dm0, dm1, dm2, dm3, dm4, dm5, dm6, dm7, dm8, dm9, dm10, dm11, dm12, dm13, dm14, dm15;
	  dm0 = dev_inter_011(xi, sj[112], &pot7);
	  dm1 = dev_inter_011(xi, sj[113], &pot7);
	  dm2 = dev_inter_011(xi, sj[114], &pot7);
	  dm3 = dev_inter_011(xi, sj[115], &pot7);
	  dm4 = dev_inter_011(xi, sj[116], &pot7);
	  dm5 = dev_inter_011(xi, sj[117], &pot7);
	  dm6 = dev_inter_011(xi, sj[118], &pot7);
	  dm7 = dev_inter_011(xi, sj[119], &pot7);
	  dm8 = dev_inter_011(xi, sj[120], &pot7);
	  dm9 = dev_inter_011(xi, sj[121], &pot7);
	  dm10 = dev_inter_011(xi, sj[122], &pot7);
	  dm11 = dev_inter_011(xi, sj[123], &pot7);
	  dm12 = dev_inter_011(xi, sj[124], &pot7);
	  dm13 = dev_inter_011(xi, sj[125], &pot7);
	  dm14 = dev_inter_011(xi, sj[126], &pot7);
	  dm15 = dev_inter_011(xi, sj[127], &pot7);
	  apot.x += (dm0.x*dm0.w) + (dm1.x*dm1.w) + (dm2.x*dm2.w) + (dm3.x*dm3.w) + (dm4.x*dm4.w) + (dm5.x*dm5.w) + (dm6.x*dm6.w) + (dm7.x*dm7.w) \
	    + (dm8.x*dm8.w) + (dm9.x*dm9.w) + (dm10.x*dm10.w) + (dm11.x*dm11.w) + (dm12.x*dm12.w) + (dm13.x*dm13.w) + (dm14.x*dm14.w) + (dm15.x*dm15.w);
	  apot.y += (dm0.y*dm0.w) + (dm1.y*dm1.w) + (dm2.y*dm2.w) + (dm3.y*dm3.w) + (dm4.y*dm4.w) + (dm5.y*dm5.w) + (dm6.y*dm6.w) + (dm7.y*dm7.w) \
	    + (dm8.y*dm8.w) + (dm9.y*dm9.w) + (dm10.y*dm10.w) + (dm11.y*dm11.w) + (dm12.y*dm12.w) + (dm13.y*dm13.w) + (dm14.y*dm14.w) + (dm15.y*dm15.w);
	  apot.z += (dm0.z*dm0.w) + (dm1.z*dm1.w) + (dm2.z*dm2.w) + (dm3.z*dm3.w) + (dm4.z*dm4.w) + (dm5.z*dm5.w) + (dm6.z*dm6.w) + (dm7.z*dm7.w) \
	    + (dm8.z*dm8.w) + (dm9.z*dm9.w) + (dm10.z*dm10.w) + (dm11.z*dm11.w) + (dm12.z*dm12.w) + (dm13.z*dm13.w) + (dm14.z*dm14.w) + (dm15.z*dm15.w);
	}

      }

      apot.w -= pot0+pot1+pot2+pot3+pot4+pot5+pot6+pot7;

      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
    }
  }


  // 2008/04/23
  __device__ float4 dev_apot(float4 xi, float4 xj, float4 apot)
  {
    float dx = xj.x - xi.x;
    float dy = xj.y - xi.y;
    float dz = xj.z - xi.z;
    float eps2 = xi.w;
    float mj   = xj.w;
    float r2   = (dx*dx + eps2) + dy*dy + dz*dz;
    float r1i  = rsqrt(r2);
    float r2i  = r1i*r1i;
    float mr1i = mj * r1i;
    float mr3i = mr1i * r2i;
    apot.x += dx * mr3i;
    apot.y += dy * mr3i;
    apot.z += dz * mr3i;
    apot.w -= mr1i;
    return (apot);
  }

  // 2008/04/23
  __global__ 
  void cunbody_kernel_tree_012(float4 *xilist,
			       float4 *xjlist,
			       float4 *apotlist,
			       unsigned int *ioffset,
			       unsigned int *joffset)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int ibegin = ioffset[bid];
    unsigned int iend   = ioffset[bid+1];
    unsigned int jbegin = joffset[bid];
    unsigned int jend   = joffset[bid+1];
    for(unsigned int ibase = ibegin; ibase < iend; ibase += NTHRE){
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

      for(unsigned int jbase = jbegin; jbase < jend; jbase += NTHRE){
	__shared__ float4 sj[NTHRE];
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	apot = dev_apot(xi, sj[0], apot); apot = dev_apot(xi, sj[1], apot); apot = dev_apot(xi, sj[2], apot); apot = dev_apot(xi, sj[3], apot);	apot = dev_apot(xi, sj[4], apot);
	apot = dev_apot(xi, sj[5], apot); apot = dev_apot(xi, sj[6], apot); apot = dev_apot(xi, sj[7], apot); apot = dev_apot(xi, sj[8], apot); apot = dev_apot(xi, sj[9], apot);
	apot = dev_apot(xi, sj[10], apot); apot = dev_apot(xi, sj[11], apot); apot = dev_apot(xi, sj[12], apot); apot = dev_apot(xi, sj[13], apot); apot = dev_apot(xi, sj[14], apot);
	apot = dev_apot(xi, sj[15], apot); apot = dev_apot(xi, sj[16], apot); apot = dev_apot(xi, sj[17], apot); apot = dev_apot(xi, sj[18], apot); apot = dev_apot(xi, sj[19], apot);
	apot = dev_apot(xi, sj[20], apot); apot = dev_apot(xi, sj[21], apot); apot = dev_apot(xi, sj[22], apot); apot = dev_apot(xi, sj[23], apot); apot = dev_apot(xi, sj[24], apot);
	apot = dev_apot(xi, sj[25], apot); apot = dev_apot(xi, sj[26], apot); apot = dev_apot(xi, sj[27], apot); apot = dev_apot(xi, sj[28], apot); apot = dev_apot(xi, sj[29], apot);
	apot = dev_apot(xi, sj[30], apot); apot = dev_apot(xi, sj[31], apot); apot = dev_apot(xi, sj[32], apot); apot = dev_apot(xi, sj[33], apot); apot = dev_apot(xi, sj[34], apot);
	apot = dev_apot(xi, sj[35], apot); apot = dev_apot(xi, sj[36], apot); apot = dev_apot(xi, sj[37], apot); apot = dev_apot(xi, sj[38], apot); apot = dev_apot(xi, sj[39], apot);
	apot = dev_apot(xi, sj[40], apot); apot = dev_apot(xi, sj[41], apot); apot = dev_apot(xi, sj[42], apot); apot = dev_apot(xi, sj[43], apot); apot = dev_apot(xi, sj[44], apot);
	apot = dev_apot(xi, sj[45], apot); apot = dev_apot(xi, sj[46], apot); apot = dev_apot(xi, sj[47], apot); apot = dev_apot(xi, sj[48], apot); apot = dev_apot(xi, sj[49], apot);
	apot = dev_apot(xi, sj[50], apot); apot = dev_apot(xi, sj[51], apot); apot = dev_apot(xi, sj[52], apot); apot = dev_apot(xi, sj[53], apot); apot = dev_apot(xi, sj[54], apot);
	apot = dev_apot(xi, sj[55], apot); apot = dev_apot(xi, sj[56], apot); apot = dev_apot(xi, sj[57], apot); apot = dev_apot(xi, sj[58], apot); apot = dev_apot(xi, sj[59], apot);
	apot = dev_apot(xi, sj[60], apot); apot = dev_apot(xi, sj[61], apot); apot = dev_apot(xi, sj[62], apot); apot = dev_apot(xi, sj[63], apot); apot = dev_apot(xi, sj[64], apot);
	apot = dev_apot(xi, sj[65], apot); apot = dev_apot(xi, sj[66], apot); apot = dev_apot(xi, sj[67], apot); apot = dev_apot(xi, sj[68], apot); apot = dev_apot(xi, sj[69], apot);
	apot = dev_apot(xi, sj[70], apot); apot = dev_apot(xi, sj[71], apot); apot = dev_apot(xi, sj[72], apot); apot = dev_apot(xi, sj[73], apot); apot = dev_apot(xi, sj[74], apot);
	apot = dev_apot(xi, sj[75], apot); apot = dev_apot(xi, sj[76], apot); apot = dev_apot(xi, sj[77], apot); apot = dev_apot(xi, sj[78], apot); apot = dev_apot(xi, sj[79], apot);
	apot = dev_apot(xi, sj[80], apot); apot = dev_apot(xi, sj[81], apot); apot = dev_apot(xi, sj[82], apot); apot = dev_apot(xi, sj[83], apot); apot = dev_apot(xi, sj[84], apot);
	apot = dev_apot(xi, sj[85], apot); apot = dev_apot(xi, sj[86], apot); apot = dev_apot(xi, sj[87], apot); apot = dev_apot(xi, sj[88], apot); apot = dev_apot(xi, sj[89], apot);
	apot = dev_apot(xi, sj[90], apot); apot = dev_apot(xi, sj[91], apot); apot = dev_apot(xi, sj[92], apot); apot = dev_apot(xi, sj[93], apot); apot = dev_apot(xi, sj[94], apot);
	apot = dev_apot(xi, sj[95], apot); apot = dev_apot(xi, sj[96], apot); apot = dev_apot(xi, sj[97], apot); apot = dev_apot(xi, sj[98], apot); apot = dev_apot(xi, sj[99], apot);
	apot = dev_apot(xi, sj[100], apot); apot = dev_apot(xi, sj[101], apot); apot = dev_apot(xi, sj[102], apot); apot = dev_apot(xi, sj[103], apot); apot = dev_apot(xi, sj[104], apot);
	apot = dev_apot(xi, sj[105], apot); apot = dev_apot(xi, sj[106], apot); apot = dev_apot(xi, sj[107], apot); apot = dev_apot(xi, sj[108], apot); apot = dev_apot(xi, sj[109], apot);
	apot = dev_apot(xi, sj[110], apot); apot = dev_apot(xi, sj[111], apot); apot = dev_apot(xi, sj[112], apot); apot = dev_apot(xi, sj[113], apot); apot = dev_apot(xi, sj[114], apot);
	apot = dev_apot(xi, sj[115], apot); apot = dev_apot(xi, sj[116], apot); apot = dev_apot(xi, sj[117], apot); apot = dev_apot(xi, sj[118], apot); apot = dev_apot(xi, sj[119], apot);
	apot = dev_apot(xi, sj[120], apot); apot = dev_apot(xi, sj[121], apot); apot = dev_apot(xi, sj[122], apot); apot = dev_apot(xi, sj[123], apot); apot = dev_apot(xi, sj[124], apot);
	apot = dev_apot(xi, sj[125], apot); apot = dev_apot(xi, sj[126], apot); apot = dev_apot(xi, sj[127], apot);
	__syncthreads();
      }

      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
    }
  }

  // 2008/04/24
  __global__ 
  void cunbody_kernel_tree_013(float4 *xilist,
			       float4 *xjlist,
			       float4 *apotlist,
			       unsigned int *ioffset,
			       unsigned int *joffset)
  {
    __syncthreads(); // iranai

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int ibegin = ioffset[bid];
    int iend   = ioffset[bid+1];
    int jbegin = joffset[bid];
    int jend   = joffset[bid+1];

    __syncthreads(); // iranai
    for(int ibase = ibegin; ibase < iend; ibase += NTHRE){

      __syncthreads(); // iranai
      float4 xi = xilist[ibase + tid];
      float4 apot = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
      __syncthreads(); // iranai
      __shared__ float4 sj[NTHRE];
      __syncthreads(); // iranai

      for(int jbase = jbegin; jbase < jend; jbase += NTHRE){
	sj[tid] = xjlist[jbase + tid];
	__syncthreads();
	for(int j=0; j<128; j++){ apot = dev_apot(xi, sj[j], apot); }
	__syncthreads();
      }

      __syncthreads(); // iranai
      if(ibase + tid < iend){
	apotlist[ibase + tid] = apot;
      }
      __syncthreads(); // iranai
    }

    __syncthreads(); // irania
  }


}; // namespace libcunbody __END__


