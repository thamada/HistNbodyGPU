#include "hip/hip_runtime.h"
#include "cutil.h"

__device__ float4
inter(float4 xj,  float4 xi,  float4 apot)
{
  float mj    = xj.w;        // Mass Mj
  float ieps2 = xi.w;        // epsilon^2
  float dx = xj.x - xi.x;    // Coordinates Xj - Xi
  float dy = xj.y - xi.y;    // Coordinates Yj - Yi
  float dz = xj.z - xi.z;    // Coordinates Zj - Zi
  float r2 = dx*dx+dy*dy+dz*dz+ieps2;
  float r1i = 1/sqrt(r2);
  float r2i = r1i * r1i;
  float mr3i = mj * r2i * r1i;
  apot.x += dx * mr3i;       // Accel AXi
  apot.y += dy * mr3i;       // Accel AYi
  apot.z += dz * mr3i;       // Accel AZi
  return (apot);
}

__global__ void
kernel(float4* g_xj,
       float* g_xi,
       float* g_fi,
       int ni,
       int nj)
{
  int i = blockIdx.x*gridDim.x+threadIdx.x;
  float4 ai = make_float4(0.0, 0.0, 0.0, 0.0);
  float4 xi;
  xi.x = g_xi[i];       // Coordinates Xi
  xi.y = g_xi[i+ni];    // Coordinates Yi
  xi.z = g_xi[i+ni*2];  // Coordinates Zi
  xi.w = g_xi[i+ni*3];  // epsilon ^ 2
  for(int j = 0; j<nj; j++)  ai = inter(g_xj[j], xi, ai);
  if(i<ni){
    g_fi[i]      = ai.x;
    g_fi[i+ni]   = ai.y;
    g_fi[i+ni*2] = ai.z;
  }
}

#define NTHRE (128)
#include "api.h"
