#include "hip/hip_runtime.h"
#include "cutil.h"

__device__ float4 
inter(float4 xj, float4 xi, float4 apot)
{
  /* �����Ͽ�5.��Ʊ�� */ 
  float mj    = xj.w;        // Mass Mj
  float ieps2 = xi.w;        // epsilon^2
  float dx = xj.x - xi.x;    // Coordinates Xj - Xi
  float dy = xj.y - xi.y;    // Coordinates Yj - Yi
  float dz = xj.z - xi.z;    // Coordinates Zj - Zi
  float r2 = dx*dx+dy*dy+dz*dz+ieps2;
  float r1i = 1/sqrt(r2);
  float r2i = r1i * r1i;
  float mr3i = mj * r2i * r1i;
  apot.x += dx * mr3i;       // Accel AXi
  apot.y += dy * mr3i;       // Accel AYi
  apot.z += dz * mr3i;       // Accel AZi
  return (apot);
}
#define NTHRE (128) // blockDim.x��Ʊ��
__global__ void
kernel(float4* g_xj,
       float* g_xi,
       float* g_fi,
       int ni,
       int nj)
{
  int tid      = threadIdx.x;
  int i = blockIdx.x*NTHRE+tid;
  float4 ai = make_float4(0.0, 0.0, 0.0, 0.0);
  float4 xi;
  xi.x = g_xi[i];
  xi.y = g_xi[i+ni];
  xi.z = g_xi[i+ni*2];
  xi.w = g_xi[i+ni*3];
  __shared__ float4 s_xj[NTHRE];
  for(int j = 0; j<nj; j+=NTHRE){
    __syncthreads();
    s_xj[tid] = g_xj[j+tid];
    __syncthreads();
    for(int js = 0; js<NTHRE; js++) ai = inter(s_xj[js], xi, ai);
  }
  if(i<ni){
    g_fi[i]      = ai.x;
    g_fi[i+ni]   = ai.y;
    g_fi[i+ni*2] = ai.z;
  }
}

#include "api.h"
